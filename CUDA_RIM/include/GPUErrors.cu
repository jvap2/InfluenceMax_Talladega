#include "../include/GPUErrors.h"

bool HandleCUDAError(hipError_t t)
{
	if (t != hipSuccess)
	{
		cout << hipGetErrorString(hipGetLastError())<<endl;//This will get the string of the error for blocking error
		cout<<t<<endl;
		return false;
	}
	return true;
}
//We can have runtime errors on the GPU, which is what the function below is used for
bool GetCUDARunTimeError()
{
	hipError_t t = hipGetLastError();
	if (t != hipSuccess)
	{
		cout << hipGetErrorString(t) << endl;
		return false;
	}
	return true;
}

bool HandleCUSparseError(hipsparseStatus_t t){
	if (t != HIPSPARSE_STATUS_SUCCESS)
	{
		cout << "CUSPARSE ERROR: " << t << endl;
		cout<< hipsparseGetErrorString(t)<<endl;
		return false;
	}
	return true;
}

bool HandleCUBLASError(hipblasStatus_t t){
	if (t != HIPBLAS_STATUS_SUCCESS)
	{
		cout << "CUBLAS ERROR: " << t << endl;
		cout<< cublasGetStatusString(t)<<endl;
		return false;
	}
	return true;
}


void printCudaMemoryUsage() {
    size_t free_byte ;
    size_t total_byte ;
    hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;

    if ( hipSuccess != cuda_status ){
        printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
        exit(1);
    }

    double free_db = (double)free_byte ;
    double total_db = (double)total_byte ;
    double used_db = total_db - free_db ;
    printf("GPU memory usage: used = %.2f, free = %.2f MB, total = %.2f MB\n",
        used_db /1024.0/1024.0, free_db /1024.0/1024.0, total_db /1024.0/1024.0);
}