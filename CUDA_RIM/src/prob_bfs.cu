#include "hip/hip_runtime.h"
#include "../include/data.h"


__host__ void Prob_BFS_Score(unsigned int* csc, unsigned int* succ, unsigned int node_size, unsigned int edge_size, unsigned int* seed_set, float threshold, string file){
    unsigned int* d_csc;
    unsigned int* d_succ;
    float* d_score;
    float* d_share_score;
    unsigned int* d_visited;
    unsigned int* d_seed_set;
    unsigned int* d_frontier;
    unsigned int* d_next_frontier;
    float* rand_numbers;
    unsigned int* rand_idx;
    float* res = new float[NUMSTRM];
    thrust::fill(thrust::host, res, res + NUMSTRM, 0);
    if(!HandleCUDAError(hipMalloc((void**)&d_csc, sizeof(unsigned int) * (node_size + 1)))){
        std::cout << "hipMalloc d_csc failed" << endl;
        exit(0);
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_succ, sizeof(unsigned int) * edge_size))){
        std::cout << "hipMalloc d_succ failed" << endl;
        exit(0);
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_score, sizeof(float) * node_size * NUMSTRM))){
        std::cout << "hipMalloc d_score failed" << endl;
        exit(0);
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_visited, sizeof(unsigned int) * node_size * NUMSTRM))){
        std::cout << "hipMalloc d_visited failed" << endl;
        exit(0);
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_share_score, sizeof(float) * node_size))){
        std::cout << "hipMalloc d_share_score failed" << endl;
        exit(0);
    }
    if(!HandleCUDAError(hipMalloc((void**)&rand_numbers, sizeof(float) * NUMSTRM))){
        std::cout << "hipMalloc rand_numbers failed" << endl;
        exit(0);
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_frontier, sizeof(unsigned int) * node_size * NUMSTRM))){
        std::cout << "hipMalloc d_frontier failed" << endl;
        exit(0);
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_next_frontier, sizeof(unsigned int) * node_size * NUMSTRM))){
        std::cout << "hipMalloc d_next_frontier failed" << endl;
        exit(0);
    }
    if(!HandleCUDAError(hipMalloc((void**)&rand_idx, sizeof(unsigned int) * node_size))){
        std::cout << "hipMalloc rand_idx failed" << endl;
        exit(0);
    }
    if(!HandleCUDAError(hipMemcpy(d_csc, csc, sizeof(unsigned int) * (node_size + 1), hipMemcpyHostToDevice))){
        std::cout << "hipMemcpy d_csc failed" << endl;
        exit(0);
    }
    if(!HandleCUDAError(hipMemcpy(d_succ, succ, sizeof(unsigned int) * edge_size, hipMemcpyHostToDevice))){
        std::cout << "hipMemcpy d_succ failed" << endl;
        exit(0);
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_seed_set, sizeof(unsigned int) * K))){
        std::cout << "hipMalloc d_seed_set failed" << endl;
        exit(0);
    }
    hipStream_t* streams = (hipStream_t*)malloc(sizeof(hipStream_t)*NUMSTRM);
    CreateStreams(NUMSTRM, streams);
    unsigned int max_blocks = Max_Blocks(TPB,NUMSTRM);
    unsigned int blocks = node_size/TPB+1;
    if(blocks > max_blocks){
        blocks = max_blocks;
    }
    thrust::fill(thrust::device, d_share_score, d_share_score + node_size, 0);
    thrust::fill(thrust::device, d_seed_set, d_seed_set + K, 0);
    unsigned int iter = 5;
    unsigned int total_visited = 0; 
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    for(int i = 0; i < iter; i++){
        thrust::fill(thrust::device, d_score, d_score + node_size * NUMSTRM, 0);
        thrust::fill(thrust::device, d_visited, d_visited + node_size * NUMSTRM, 0);
        thrust::fill(thrust::device, rand_numbers, rand_numbers + NUMSTRM, 0);
        hiprandGenerator_t gen;
        hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
        srand(time(0));
        int rand_seed = rand();
        hiprandSetPseudoRandomGeneratorSeed(gen, rand_seed);
        hiprandGenerateUniform(gen, rand_numbers, NUMSTRM+1);
        hiprandDestroyGenerator(gen);
        thrust::transform(thrust::device, rand_numbers, rand_numbers + NUMSTRM, rand_numbers, [=] __device__ (float rand_num) { return rand_num*node_size; });
        for(int j = 0; j<NUMSTRM;j++){
            unsigned int* frontier_j = d_frontier + j * node_size;
            thrust::transform(thrust::device, rand_numbers+j, rand_numbers + j+1, frontier_j, [=] __device__ (float rand_num) {
                int index = static_cast<int>(rand_num);
                if (index >= 0 && index < node_size) {
                    return 1;
                } else {
                    return 0;
                }
            });
        }
        std::cout << "iter: " << i << endl;
        unsigned int level = 1;
        do{
            for(int j = 0; j<NUMSTRM;j++){
                unsigned int* d_visited_j = d_visited + j * node_size;
                float* d_score_j = d_score + j * node_size;
                unsigned int* frontier_j = d_frontier + j * node_size;
                unsigned int* next_j = d_next_frontier + j * node_size;
                Prob_BFS_Score_Kernel<<<blocks, TPB, 0, streams[j]>>>(d_csc, d_succ, node_size, edge_size, d_score_j, d_visited_j, frontier_j, next_j, threshold, level);
                if(!HandleCUDAError(hipStreamSynchronize(streams[j]))){
                    std::cout << "hipStreamSynchronize failed" << endl;
                    exit(0);
                }
                res[j]=thrust::reduce(thrust::device.on(streams[j]), d_visited_j, d_visited_j + node_size, 0);
                // cout<<"res[j]: "<<res[j]<<endl;
                // cout<<"Node size: "<<node_size<<endl;   
            }
            thrust::copy(thrust::device, d_next_frontier, d_next_frontier + node_size * NUMSTRM, d_frontier);
            thrust::fill(thrust::device, d_next_frontier, d_next_frontier + node_size * NUMSTRM, 0);
            total_visited = thrust::reduce(thrust::host, res, res + NUMSTRM, 0);
            level++;
        }while(total_visited < NUMSTRM*node_size && level<=10);
        //Collect the sum of the scores and place into shared score
        for(int k = 0; k<NUMSTRM;k++){
            float* d_score_k = d_score + k * node_size;
            // thrust::transform(thrust::device.on(streams[k]), d_score_k, d_score_k + node_size, d_score_k, [=] __device__ (float score) { return exp(score); });
            thrust::transform(thrust::device.on(streams[k]), d_score_k, d_score_k + node_size, d_share_score, d_share_score, thrust::plus<float>());
        }

    }
    thrust::sequence(thrust::device, rand_idx, rand_idx + node_size, 0);
    // Sort d_share_score in descending order
    thrust::sort_by_key(thrust::device, d_share_score, d_share_score+node_size, rand_idx, thrust::greater<float>());
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Time taken: " << milliseconds << "ms" << endl;
    unsigned int* h_rand_idx = new unsigned int[node_size];
    if(!HandleCUDAError(hipMemcpy(h_rand_idx, rand_idx, sizeof(unsigned int)*node_size, hipMemcpyDeviceToHost))){
        std::cout<<"Error copying rand_idx to host"<<endl;
    }
    for(int i = 0; i < K; i++){
        seed_set[i] = h_rand_idx[i];
    }
    delete[] h_rand_idx;
    float* debug = new float[node_size];
    if(!HandleCUDAError(hipMemcpy(debug, d_share_score, sizeof(float)*node_size, hipMemcpyDeviceToHost))){
        std::cout<<"Error copying d_share_score to host"<<endl;
    }
    if(!HandleCUDAError(hipFree(d_csc))){
        std::cout << "hipFree d_csc failed" << endl;
        exit(0);
    }
    if(!HandleCUDAError(hipFree(d_succ))){
        std::cout << "hipFree d_succ failed" << endl;
        exit(0);
    }
    if(!HandleCUDAError(hipFree(d_score))){
        std::cout << "hipFree d_score failed" << endl;
        exit(0);
    }
    if(!HandleCUDAError(hipFree(d_visited))){
        std::cout << "hipFree d_visited failed" << endl;
        exit(0);
    }
    if(!HandleCUDAError(hipFree(d_share_score))){
        std::cout << "hipFree d_share_score failed" << endl;
        exit(0);
    }
    if(!HandleCUDAError(hipFree(d_seed_set))){
        std::cout << "hipFree d_seed_set failed" << endl;
        exit(0);
    }
    if(!HandleCUDAError(hipFree(d_frontier))){
        std::cout << "hipFree d_frontier failed" << endl;
        exit(0);
    }
    if(!HandleCUDAError(hipFree(d_next_frontier))){
        std::cout << "hipFree d_next_frontier failed" << endl;
        exit(0);
    }
    if(!HandleCUDAError(hipFree(rand_numbers))){
        std::cout << "hipFree rand_numbers failed" << endl;
        exit(0);
    }
    if(!HandleCUDAError(hipFree(rand_idx))){
        std::cout << "hipFree rand_idx failed" << endl;
        exit(0);
    }
    free(streams);
    delete[] res;
    if(!HandleCUDAError(hipDeviceReset())){
        std::cout << "hipDeviceReset failed" << endl;
        exit(0);
    }
    for(int i = 0; i < K; i++){
        cout<<seed_set[i]<<endl;
        cout<<debug[i]<<endl;
    }
    delete[] debug;
}




__global__ void Prob_BFS_Score_Kernel(unsigned int* d_csc, unsigned int* d_succ, unsigned int node_size, unsigned int edge_size, float* d_score, unsigned int* d_visited,
unsigned int* frontier, unsigned int* next_frontier, float threshold, int level)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    for(int i = idx; i < node_size; i += blockDim.x * gridDim.x){
        if(frontier[i]==1){
            //This means it is in this current frontier
            //We need to go to the next frontier
            frontier[i] = 0;
            unsigned int start = d_csc[i];
            unsigned int end = d_csc[i+1];
            for(int j = start; j < end; j++){
                unsigned int neighbor = d_succ[j];
                if(d_visited[neighbor]==0){
                    //This means it has not been visited
                    //We need to add it to the next frontier
                    d_visited[neighbor] = 1;
                    next_frontier[neighbor] = 1;
                    if(level <=1){
                        d_score[neighbor] += exp(-(powf(log(1-threshold),2.0f))/((2/3)*powf(log(1-threshold),2.0f)+(2/3)*log(1-threshold)+1));
                    }
                    else
                        d_score[neighbor] += exp(-(powf(log(1-threshold),2.0f))/((2/3)*powf(log(1-threshold),2.0f)+(2/3)*log(1-threshold)+1))*d_score[i];
                }
            }
        }
    }
}



__host__ void  RIM_rand_Mart_BFS(unsigned int* csc, unsigned int* succ, unsigned int node_size, unsigned int edge_size, unsigned int* seed_set, float threshold, string file){
    float damping_factor =.3;
    hipDeviceProp_t prop;
    int device;
    hipGetDevice(&device);  // Get the current device
    hipGetDeviceProperties(&prop, device);  // Get the properties of the device

    int maxActiveBlocksPerMultiprocessor = prop.maxThreadsPerMultiProcessor / TPB;
    int maxActiveBlocks = prop.multiProcessorCount * maxActiveBlocksPerMultiprocessor;
    int blocks_per_stream = maxActiveBlocks/NUMSTRM;

    printf("Max active blocks: %d\n", maxActiveBlocks);
    hipStream_t* streams = (hipStream_t*)malloc(sizeof(hipStream_t)*NUMSTRM);
    for(int i = 0; i < NUMSTRM; i++){
        if(!HandleCUDAError(hipStreamCreate(&streams[i]))){
            std::cout<<"Error creating stream number "<<i<<endl;
        }
    }
    unsigned int num_walker = 1;
    unsigned int epochs=30;
    unsigned int* d_csc;
    unsigned int* d_succ;
    float* d_vec; //we will use the seed set as the PR vector and then transfer the top k to the actual seed set
    float* d_res;
    float* d_track_val;
    float* vec = new float[NUMSTRM*node_size];
    float* values = new float[NUMSTRM*edge_size];
    float* res = new float[NUMSTRM*node_size]; 
    float* tol = new float[NUMSTRM];
    float* temp_sum = new float[NUMSTRM];
    float* sum = new float[NUMSTRM];
    float* l2_norm_d_res = new float[NUMSTRM];
    float* l2_norm_rand_vec_init = new float[NUMSTRM];
    thrust::fill(sum, sum+NUMSTRM, 0.0f);
    thrust::fill(tol,tol+NUMSTRM, 100.0f);
    thrust::fill(res, res+NUMSTRM*node_size, 0.0f);
    thrust::fill(vec, vec+NUMSTRM*node_size, 1.0f/node_size);
    thrust::fill(values, values+NUMSTRM*edge_size, 1.0f);
    unsigned int* d_check;
    if(!HandleCUDAError(hipMalloc((void**)&d_csc, sizeof(unsigned int)*(node_size+1)))){
        std::cout<<"Error allocating memory for d_csc"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_succ, sizeof(unsigned int)*(edge_size)))){
        std::cout<<"Error allocating memory for d_succ"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_vec, sizeof(float)*node_size*NUMSTRM))){
        std::cout<<"Error allocating memory for d_seed_set"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_res, sizeof(float)*node_size*NUMSTRM))){
        std::cout<<"Error allocating memory for d_res"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_track_val, sizeof(float)*node_size*NUMSTRM))){
        std::cout<<"Error allocating memory for d_res"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_check, sizeof(unsigned int)*NUMSTRM*node_size))){
        std::cout<<"Error allocating memory for d_check"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_csc, csc, sizeof(unsigned int)*node_size, hipMemcpyHostToDevice))){
        std::cout<<"Error copying csc to device"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_succ, succ, sizeof(unsigned int)*edge_size, hipMemcpyHostToDevice))){
        std::cout<<"Error copying succ to device"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_vec, vec, sizeof(float)*node_size*NUMSTRM, hipMemcpyHostToDevice))){
        std::cout<<"Error copying vec to device"<<endl;
    }
    delete[] vec;
    if(!HandleCUDAError(hipMemcpy(d_res, res, sizeof(float)*node_size*NUMSTRM, hipMemcpyHostToDevice))){
        std::cout<<"Error copying res to device"<<endl;
    }
    delete[] res;

    
    float* d_values;
    if(!HandleCUDAError(hipMalloc((void**)&d_values, sizeof(float)*(edge_size*NUMSTRM)))){
        std::cout<<"Error allocating memory for d_values"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_values, values, sizeof(float)*edge_size*NUMSTRM, hipMemcpyHostToDevice))){
        std::cout<<"Error copying values to device"<<endl;
    }
    // delete[] values;


    float* rand_init;
    if(!HandleCUDAError(hipMalloc((void**)&rand_init, NUMSTRM*num_walker*sizeof(float)))){
        std::cout<<"Error allocating memory for rand_frog"<<endl;
    }
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);

    /*Now, we have the random numbers generated*/
    float* rand_vec_init;
    float* h_rand_vec_init = new float[node_size*NUMSTRM];
    thrust::fill(h_rand_vec_init, h_rand_vec_init+node_size*NUMSTRM, 0.0f);
    if(!HandleCUDAError(hipMalloc((void**)&rand_vec_init, sizeof(float)*node_size*NUMSTRM))){
        std::cout<<"Error allocating memory for rand_vec_init"<<endl;
    } 
    if(!HandleCUDAError(hipMemcpy(rand_vec_init, h_rand_vec_init, sizeof(float)*node_size*NUMSTRM, hipMemcpyHostToDevice))){
        std::cout<<"Error copying h_rand_vec_init to device"<<endl;
    }

    float* store_stream_res;
    float* h_store_stream_res = new float[node_size*NUMSTRM];
    thrust::fill(h_store_stream_res, h_store_stream_res+node_size*NUMSTRM, 0.0f);
    if(!HandleCUDAError(hipMalloc((void**)&store_stream_res, sizeof(float)*node_size*NUMSTRM))){
        std::cout<<"Error allocating memory for store_stream_res"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(store_stream_res, h_store_stream_res, sizeof(float)*node_size*NUMSTRM, hipMemcpyHostToDevice))){
        std::cout<<"Error copying h_store_stream_res to device"<<endl;
    }
    delete[] h_store_stream_res;
    float* rand_numbers;
    if (!HandleCUDAError(hipMalloc((void**)&rand_numbers, sizeof(float) * edge_size*NUMSTRM))) {
        std::cout << "Error allocating memory for rand_numbers" << endl;
    }
    printCudaMemoryUsage();
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    float tol_thresh=1;
    epochs = 30*(K/NUMSTRM+1);
    for(int k = 0; k < epochs; k++){
        // std::cout<<"Epoch "<<k<<endl;
        thrust::fill(tol,tol+NUMSTRM, 100.0f);
        thrust::fill(sum, sum+NUMSTRM, 0.0f);
        thrust::fill(temp_sum, temp_sum+NUMSTRM, 0.0f);
        int while_count = 0;
        while_count=0;
        srand(time(0));
        int rand_seed = rand();
        hiprandSetPseudoRandomGeneratorSeed(gen, rand_seed);
        hiprandGenerateUniform(gen, rand_init, num_walker*NUMSTRM);
        for(int i = 0; i < NUMSTRM; i++){
            //Initialize the random vector
            float* rand_init_i = rand_init + i*num_walker;
            float* rand_vec_init_i = rand_vec_init + i*node_size;
            unsigned int* d_check_i = d_check + i*node_size;
            thrust::fill(thrust::device.on(streams[i]), rand_vec_init_i, rand_vec_init_i+node_size, 0.0f);
            thrust::fill(thrust::device.on(streams[i]),d_check_i, d_check_i+node_size, 0);
            thrust::fill(thrust::device.on(streams[i]),d_res+i*node_size, d_res+i*node_size+node_size, 0.0f);
            thrust::fill(thrust::device.on(streams[i]),d_track_val+i*node_size, d_track_val+i*node_size+node_size, 0.0f);
            Init_Random<<<blocks_per_stream, TPB,0,streams[i]>>>(rand_vec_init_i, rand_init_i, node_size, num_walker);
            if(!HandleCUDAError(hipStreamSynchronize(streams[i]))){
                std::cout<<"Error synchronizing device at Init Random for Stream "<<i<<endl;
            }
            float* rand_numbers_i = rand_numbers + i*edge_size;
            float* d_values_i = d_values + i*edge_size;
            hiprandGenerator_t gen;
            hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
            srand(time(0));
            int rand_seed = rand();
            hiprandSetPseudoRandomGeneratorSeed(gen, rand_seed);
            hiprandGenerateUniform(gen, rand_numbers_i, edge_size);
            hiprandDestroyGenerator(gen);
            thrust::fill(thrust::device.on(streams[i]), d_values_i, d_values_i+edge_size, 1.0f);
            thrust::transform(thrust::device.on(streams[i]), rand_numbers_i, rand_numbers_i+edge_size, d_values_i, d_values_i, [threshold] __device__ (float x, float y) { return eval_values(x,y,threshold); });
        }
        bool check = true;
        while(check && while_count < MAX_WHILE){
            while_count++;
            // cout<<"While count: "<<while_count<<endl;
            float level_thresh = exp(-(powf(log(1-threshold),2.0f))/((2/3)*powf(log(1-threshold),2.0f)+(2/3)*log(1-threshold)+1));
            for(int i = 0; i < NUMSTRM; i++){
                //Perform the first iteration of the algorithm
                if(tol[i] > 0){
                    float* rand_vec_init_i = rand_vec_init + i*node_size;
                    float* d_res_i = d_res + i*node_size;
                    float* d_values_i = d_values + i*edge_size;
                    float* d_track_i = d_track_val + i*node_size;
                    unsigned int* d_check_i = d_check + i*node_size;
                    sparseCSRMat_Vec_Mult_Mart_BFS<unsigned int><<<blocks_per_stream, TPB,0,streams[i]>>>(d_csc, d_succ, d_values_i, rand_vec_init_i, d_res_i,d_check_i, threshold, node_size);  
                    if(!HandleCUDAError(hipStreamSynchronize(streams[i]))){
                        std::cout<<"Error synchronizing device at sparseCSRMat_Vec_Mult for stream "<<i<<endl;
                    }
                    // thrust::transform(thrust::device.on(streams[i]), d_res_i, d_res_i+node_size, d_res_i, [=] __device__ (float x) { return x*level_thresh; });
                    Copy<float><<<blocks_per_stream, TPB,0,streams[i]>>>(d_res_i, rand_vec_init_i, node_size);
                    if(!HandleCUDAError(hipStreamSynchronize(streams[i]))){
                        std::cout<<"Error synchronizing device at Copy for stream "<<i<<endl;
                    }
                    Float_VectAdd<<<blocks_per_stream, TPB,0,streams[i]>>>(d_track_i, rand_vec_init_i, node_size);
                    if(!HandleCUDAError(hipStreamSynchronize(streams[i]))){
                        std::cout<<"Error synchronizing device at Float_VectAdd for stream "<<i<<endl;
                    }
                    sum[i] = thrust::reduce(thrust::device.on(streams[i]), d_check_i, d_check_i+node_size);
                    tol[i] = sum[i]-temp_sum[i];
                    temp_sum[i] = sum[i];
                    // cout<<"Tol: "<<tol[i]<<endl;
                    // cout<<"Level Thresh: "<<level_thresh<<endl;
                    // cout<<"sum[i]: "<<sum[i]<<endl; 
                    thrust::fill(thrust::device.on(streams[i]), d_res_i, d_res_i+node_size, 0.0f);
                }
            }
            check = false;
            for(int i=0;i<NUMSTRM;i++){
                if(tol[i] > 0){
                    check = true;
                }
            }
        }
        for(int i = 0; i<NUMSTRM;i++){
            float* rand_vec_init_i = d_track_val + i*node_size;
            float* store_stream_res_i = store_stream_res + i*node_size;
            //Take the softmax of rand_vec_init_i
            sum[i] = thrust::reduce(thrust::device.on(streams[i]), rand_vec_init_i, rand_vec_init_i+node_size);
            float temp = sum[i];
            // cout<<"Sum: "<<sum[i]<<endl;
            if(sum[i]>0){
                thrust::transform(thrust::device.on(streams[i]), rand_vec_init_i, rand_vec_init_i+node_size, rand_vec_init_i, [=] __device__ (float x) { return x/temp; });
                Float_VectAdd<<<blocks_per_stream, TPB,0,streams[i]>>>(store_stream_res_i,rand_vec_init_i, node_size);
                if(!HandleCUDAError(hipStreamSynchronize(streams[i]))){
                    std::cout<<"Error synchronizing device for Float_VectAdd at stream "<<i<<endl;
                }
            }
            thrust::fill(thrust::device.on(streams[i]), rand_vec_init_i, rand_vec_init_i+node_size, 0);
        }
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hiprandDestroyGenerator(gen);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout<<"Time taken: "<<milliseconds<<endl;
    Save_Data(file,blocks_per_stream, milliseconds, damping_factor, threshold, epochs);
    if(!HandleCUDAError(hipFree(d_csc))){
        std::cout<<"Error freeing d_csc"<<endl;
    }
    if(!HandleCUDAError(hipFree(d_succ))){
        std::cout<<"Error freeing d_succ"<<endl;
    }
    if(!HandleCUDAError(hipFree(d_vec))){
        std::cout<<"Error freeing d_vec"<<endl;
    }
    if(!HandleCUDAError(hipFree(d_res))){
        std::cout<<"Error freeing d_res"<<endl;
    }
    if(!HandleCUDAError(hipFree(d_values))){
        std::cout<<"Error freeing d_values"<<endl;
    }
    unsigned int* rand_idx;
    unsigned int* h_rand_idx = new unsigned int[node_size];
    thrust::fill(h_rand_idx, h_rand_idx+node_size, 0);
    if(!HandleCUDAError(hipMalloc((void**)&rand_idx, sizeof(unsigned int)*node_size))){
        std::cout<<"Error allocating memory for rand_idx"<<endl;
    }
    thrust::sequence(thrust::device, rand_idx, rand_idx+node_size);
    //Take the sum of the vectors and then sort them
    for(int i = 1; i<NUMSTRM;i++){
        thrust::transform(thrust::device, store_stream_res, store_stream_res+node_size, store_stream_res+i*node_size, store_stream_res, thrust::plus<float>());
    }
    thrust::sort_by_key(thrust::device, store_stream_res, store_stream_res+node_size, rand_idx, thrust::greater<float>());
    //Get the top k indexes
    float* h_store_stream_res_fin = new float[node_size*NUMSTRM];
    if(!HandleCUDAError(hipMemcpy(h_store_stream_res_fin, store_stream_res, sizeof(float)*node_size*NUMSTRM, hipMemcpyDeviceToHost))){
        std::cout<<"Error copying store_stream_res to host"<<endl;
    }
    
    if(!HandleCUDAError(hipMemcpy(h_rand_idx, rand_idx, sizeof(unsigned int)*K, hipMemcpyDeviceToHost))){
        std::cout<<"Error copying rand_idx to host"<<endl;
    }
    for(int i = 0; i < K; i++){
        seed_set[i] = h_rand_idx[i];
        cout<<seed_set[i]<<endl;
        cout<<h_store_stream_res_fin[i]<<endl;
    }
    if(!HandleCUDAError(hipFree(rand_idx))){
        std::cout<<"Error freeing rand_idx"<<endl;
    }
    delete[] h_rand_idx;

    for(int i = 0; i<NUMSTRM;i++){
        if(!HandleCUDAError(hipStreamDestroy(streams[i]))){
            std::cout<<"Error destroying stream number "<<i<<endl;
        }
    }
    if(!HandleCUDAError(hipFree(store_stream_res))){
        std::cout<<"Error freeing store_stream_res"<<endl;
    }
    if(!HandleCUDAError(hipFree(rand_numbers))){
        std::cout<<"Error freeing rand_numbers"<<endl;
    }
    if(!HandleCUDAError(hipFree(rand_vec_init))){
        std::cout<<"Error freeing rand_vec_init"<<endl;
    }
    if(!HandleCUDAError(hipFree(rand_init))){
        std::cout<<"Error freeing rand_init"<<endl;
    }
    if(!HandleCUDAError(hipDeviceReset())){
        std::cout<<"Error resetting device"<<endl;
    }
    delete[] h_rand_vec_init;
    delete[] l2_norm_rand_vec_init;
    delete[] l2_norm_d_res;
    delete[] sum;
    delete[] tol;
    delete[] values;

}


__host__ void  RIM_rand_Mart_BFS_v2(unsigned int* csc, unsigned int* succ, unsigned int node_size, unsigned int edge_size, unsigned int* seed_set, float threshold, string file){
    float damping_factor =.3;
    hipDeviceProp_t prop;
    int device;
    hipGetDevice(&device);  // Get the current device
    hipGetDeviceProperties(&prop, device);  // Get the properties of the device

    int maxActiveBlocksPerMultiprocessor = prop.maxThreadsPerMultiProcessor / TPB;
    int maxActiveBlocks = prop.multiProcessorCount * maxActiveBlocksPerMultiprocessor;
    int blocks_per_stream = maxActiveBlocks/NUMSTRM;

    printf("Max active blocks: %d\n", maxActiveBlocks);
    hipStream_t* streams = (hipStream_t*)malloc(sizeof(hipStream_t)*NUMSTRM);
    for(int i = 0; i < NUMSTRM; i++){
        if(!HandleCUDAError(hipStreamCreate(&streams[i]))){
            std::cout<<"Error creating stream number "<<i<<endl;
        }
    }
    unsigned int num_walker = 1;
    unsigned int epochs;
    unsigned int* d_csc;
    unsigned int* d_succ;
    unsigned int* count = new unsigned int[node_size];
    float* d_vec; //we will use the seed set as the PR vector and then transfer the top k to the actual seed set
    float* d_res;
    float* d_track_val;
    float* vec = new float[NUMSTRM*node_size];
    float* values = new float[NUMSTRM*edge_size];
    float* res = new float[NUMSTRM*node_size]; 
    float* tol = new float[NUMSTRM];
    float* temp_sum = new float[NUMSTRM];
    float* sum = new float[NUMSTRM];
    float* l2_norm_d_res = new float[NUMSTRM];
    float* l2_norm_rand_vec_init = new float[NUMSTRM];
    float* d_penality;
    float* d_values_temp;
    unsigned int* max_index = new unsigned int[NUMSTRM]; 
    float* penalty_sum = new float[NUMSTRM];
    thrust::fill(count, count+node_size, 0);
    thrust::fill(sum, sum+NUMSTRM, 0.0f);
    thrust::fill(tol,tol+NUMSTRM, 100.0f);
    thrust::fill(res, res+NUMSTRM*node_size, 0.0f);
    thrust::fill(vec, vec+NUMSTRM*node_size, 1.0f/node_size);
    thrust::fill(values, values+NUMSTRM*edge_size, 1.0f);
    thrust::fill(penalty_sum, penalty_sum+NUMSTRM, 0.0f);
    thrust::fill(max_index, max_index+NUMSTRM, 0);
    unsigned int* d_check;
    unsigned int* d_max;
    if(!HandleCUDAError(hipMalloc((void**)&d_csc, sizeof(unsigned int)*(node_size+1)))){
        std::cout<<"Error allocating memory for d_csc"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_succ, sizeof(unsigned int)*(edge_size)))){
        std::cout<<"Error allocating memory for d_succ"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_vec, sizeof(float)*node_size*NUMSTRM))){
        std::cout<<"Error allocating memory for d_seed_set"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_res, sizeof(float)*node_size*NUMSTRM))){
        std::cout<<"Error allocating memory for d_res"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_track_val, sizeof(float)*node_size*NUMSTRM))){
        std::cout<<"Error allocating memory for d_res"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_check, sizeof(unsigned int)*NUMSTRM*node_size))){
        std::cout<<"Error allocating memory for d_check"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_penality, sizeof(float)*node_size*NUMSTRM))){
        std::cout<<"Error allocating memory for d_penality"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_max, sizeof(unsigned int)*NUMSTRM))){
        std::cout<<"Error allocating memory for d_max"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_values_temp, sizeof(float)*edge_size*NUMSTRM))){
        std::cout<<"Error allocating memory for d_values_temp"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_csc, csc, sizeof(unsigned int)*(node_size+1), hipMemcpyHostToDevice))){
        std::cout<<"Error copying csc to device"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_succ, succ, sizeof(unsigned int)*edge_size, hipMemcpyHostToDevice))){
        std::cout<<"Error copying succ to device"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_vec, vec, sizeof(float)*node_size*NUMSTRM, hipMemcpyHostToDevice))){
        std::cout<<"Error copying vec to device"<<endl;
    }
    delete[] vec;
    if(!HandleCUDAError(hipMemcpy(d_res, res, sizeof(float)*node_size*NUMSTRM, hipMemcpyHostToDevice))){
        std::cout<<"Error copying res to device"<<endl;
    }
    delete[] res;

    
    float* d_values;
    if(!HandleCUDAError(hipMalloc((void**)&d_values, sizeof(float)*(edge_size*NUMSTRM)))){
        std::cout<<"Error allocating memory for d_values"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_values, values, sizeof(float)*edge_size*NUMSTRM, hipMemcpyHostToDevice))){
        std::cout<<"Error copying values to device"<<endl;
    }
    // delete[] values;


    float* rand_init;
    if(!HandleCUDAError(hipMalloc((void**)&rand_init, NUMSTRM*num_walker*sizeof(float)))){
        std::cout<<"Error allocating memory for rand_frog"<<endl;
    }
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);

    /*Now, we have the random numbers generated*/
    float* rand_vec_init;
    float* h_rand_vec_init = new float[node_size*NUMSTRM];
    thrust::fill(h_rand_vec_init, h_rand_vec_init+node_size*NUMSTRM, 0.0f);
    if(!HandleCUDAError(hipMalloc((void**)&rand_vec_init, sizeof(float)*node_size*NUMSTRM))){
        std::cout<<"Error allocating memory for rand_vec_init"<<endl;
    } 
    if(!HandleCUDAError(hipMemcpy(rand_vec_init, h_rand_vec_init, sizeof(float)*node_size*NUMSTRM, hipMemcpyHostToDevice))){
        std::cout<<"Error copying h_rand_vec_init to device"<<endl;
    }

    float* store_stream_res;
    float* h_store_stream_res = new float[node_size*NUMSTRM];
    if(!HandleCUDAError(hipMalloc((void**)&store_stream_res, sizeof(float)*node_size*NUMSTRM))){
        std::cout<<"Error allocating memory for store_stream_res"<<endl;
    }
    thrust::fill(thrust::device, store_stream_res, store_stream_res+node_size*NUMSTRM, 0.0f);
    delete[] h_store_stream_res;
    float* rand_numbers;
    if (!HandleCUDAError(hipMalloc((void**)&rand_numbers, sizeof(float) * edge_size*NUMSTRM))) {
        std::cout << "Error allocating memory for rand_numbers" << endl;
    }
    thrust::fill(thrust::device, d_penality, d_penality+node_size, 0.0f);
    printCudaMemoryUsage();
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    float tol_thresh=1;
    epochs = 500*(K/NUMSTRM+1);
    for(int i = 0; i<NUMSTRM;i++){
        thrust::fill(thrust::device.on(streams[i]), d_values+i*edge_size, d_values+i*edge_size+edge_size, 1.0f/(1.0f*edge_size));
        thrust::fill(thrust::device.on(streams[i]), d_values_temp+i*edge_size, d_values_temp+i*edge_size+edge_size, 1.0f/(1.0f*edge_size));
    }
    for(int k = 0; k < epochs; k++){
        // std::cout<<"Epoch "<<k<<endl;
        thrust::fill(tol,tol+NUMSTRM, 100.0f);
        thrust::fill(sum, sum+NUMSTRM, 0.0f);
        thrust::fill(temp_sum, temp_sum+NUMSTRM, 0.0f);
        srand(time(0));
        int rand_seed = rand();
        hiprandSetPseudoRandomGeneratorSeed(gen, rand_seed);
        hiprandGenerateUniform(gen, rand_init, num_walker*NUMSTRM);
        for(int i = 0; i < NUMSTRM; i++){
            //Initialize the random vector
            float* rand_init_i = rand_init + i*num_walker;
            float* rand_vec_init_i = rand_vec_init + i*node_size;
            unsigned int* d_check_i = d_check + i*node_size;
            thrust::fill(thrust::device.on(streams[i]), rand_vec_init_i, rand_vec_init_i+node_size, 0.0f);
            thrust::fill(thrust::device.on(streams[i]),d_check_i, d_check_i+node_size, 0);
            thrust::fill(thrust::device.on(streams[i]),d_res+i*node_size, d_res+i*node_size+node_size, 0.0f);
            thrust::fill(thrust::device.on(streams[i]),d_track_val+i*node_size, d_track_val+i*node_size+node_size, 0.0f);
            Init_Random<<<blocks_per_stream, TPB,0,streams[i]>>>(rand_vec_init_i, rand_init_i, node_size, num_walker);
            if(!HandleCUDAError(hipStreamSynchronize(streams[i]))){
                std::cout<<"Error synchronizing device at Init Random for Stream "<<i<<endl;
            }
            thrust::copy(thrust::device.on(streams[i]), rand_vec_init_i, rand_vec_init_i+node_size, d_check_i);
            float* rand_numbers_i = rand_numbers + i*edge_size;
            float* d_values_i = d_values + i*edge_size;
            // hiprandGenerator_t gen;
            // hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
            // srand(time(0));
            // int rand_seed = rand();
            // hiprandSetPseudoRandomGeneratorSeed(gen, rand_seed);
            // hiprandGenerateUniform(gen, rand_numbers_i, edge_size);
            // hiprandDestroyGenerator(gen);
            thrust::copy(thrust::device.on(streams[i]), d_values_temp+i*edge_size, d_values_temp+i*edge_size+edge_size, d_values_i);
            // thrust::transform(thrust::device.on(streams[i]), rand_numbers_i, rand_numbers_i+edge_size, d_values_i, d_values_i, [threshold] __device__ (float x, float y) { return eval_values_v2(x,y,threshold); });
        }
        int while_count = 0;
        bool check = true;
        while(check){
            while_count++;
            // cout<<"While count: "<<while_count<<endl;
            // float level_thresh = exp(-(powf(log(1-threshold),2.0f))/((2/3)*powf(log(1-threshold),2.0f)+(2/3)*log(1-threshold)+1));
            for(int i = 0; i < NUMSTRM; i++){
                //Perform the first iteration of the algorithm
                if(tol[i] > 0){
                    float* rand_vec_init_i = rand_vec_init + i*node_size;
                    float* d_res_i = d_res + i*node_size;
                    float* d_values_i = d_values + i*edge_size;
                    float* d_track_i = d_track_val + i*node_size;
                    unsigned int* d_check_i = d_check + i*node_size;
                    float* d_penality_i = d_penality + i*node_size;
                    sparseCSRMat_Vec_Mult_Mart_BFS<unsigned int><<<blocks_per_stream, TPB,0,streams[i]>>>(d_csc, d_succ, d_values_i, rand_vec_init_i, d_res_i,d_check_i, threshold, node_size);  
                    if(!HandleCUDAError(hipStreamSynchronize(streams[i]))){
                        std::cout<<"Error synchronizing device at sparseCSRMat_Vec_Mult for stream "<<i<<endl;
                        exit(0);
                    }
                    // thrust::transform(thrust::device.on(streams[i]), d_res_i, d_res_i+node_size, d_res_i, [=] __device__ (float x) { return x*level_thresh; });
                    Copy<float><<<blocks_per_stream, TPB,0,streams[i]>>>(d_res_i, rand_vec_init_i, node_size);
                    if(!HandleCUDAError(hipStreamSynchronize(streams[i]))){
                        std::cout<<"Error synchronizing device at Copy for stream "<<i<<endl;
                        exit(0);
                    }
                    Float_VectAdd<<<blocks_per_stream, TPB,0,streams[i]>>>(d_track_i, rand_vec_init_i, node_size);
                    if(!HandleCUDAError(hipStreamSynchronize(streams[i]))){
                        std::cout<<"Error synchronizing device at Float_VectAdd for stream "<<i<<endl;
                        exit(0);
                    }
                    thrust::device_ptr<unsigned int> d_check_ptr(d_check_i);
                    sum[i] = thrust::reduce(thrust::device.on(streams[i]), d_check_ptr, d_check_ptr+node_size);
                    tol[i] = sum[i]-temp_sum[i];
                    temp_sum[i] = sum[i];
                    thrust::fill(thrust::device.on(streams[i]), d_res_i, d_res_i+node_size, 0.0f);
                }
            }
            check = false;
            for(int i=0;i<NUMSTRM;i++){
                if(tol[i] > 0){
                    check = true;
                }
            }
        }
        for(int i = 0; i<NUMSTRM;i++){
            float* rand_vec_init_i = d_track_val + i*node_size;
            float* r_v_i = rand_vec_init + i*node_size;
            float* store_stream_res_i = store_stream_res + i*node_size;
            float* d_penality_i = d_penality + i*node_size;
            //Take the softmax of rand_vec_init_i
            thrust::copy(thrust::device.on(streams[i]), rand_vec_init_i, rand_vec_init_i+node_size, d_penality_i);
            sum[i] = thrust::reduce(thrust::device.on(streams[i]), d_penality_i, d_penality_i+node_size);
            float temp = sum[i];
            // cout<<"Sum: "<<sum[i]<<endl;
            if(sum[i]>0){
                Float_VectAdd<<<blocks_per_stream, TPB,0,streams[i]>>>(store_stream_res_i,rand_vec_init_i, node_size);
                if(!HandleCUDAError(hipStreamSynchronize(streams[i]))){
                    std::cout<<"Error synchronizing device for Float_VectAdd at stream "<<i<<endl;
                }
                //Find the index of the maximum element in the vector
                float* iter = thrust::max_element(thrust::device.on(streams[i]), rand_vec_init_i, rand_vec_init_i+node_size);
                max_index[i] = iter - rand_vec_init_i;
                count[max_index[i]]++;
            }
            unsigned int nmstrm = NUMSTRM;
            // thrust::transform(thrust::device.on(streams[i]), d_penality, d_penality+node_size, d_penality, [=] __device__ (float x) { return (x/nmstrm); });
            thrust::fill(thrust::device.on(streams[i]), rand_vec_init_i, rand_vec_init_i+node_size, 0);
        }
        if(!HandleCUDAError(hipMemcpy(d_max, max_index, sizeof(unsigned int)*NUMSTRM, hipMemcpyHostToDevice))){
            std::cout<<"Error copying max_index to device"<<endl;
        }
        for(int i=0; i<NUMSTRM;i++){
            float* d_values_temp_i = d_values_temp + i*edge_size;
            Zero_Rows_Max_Idx<<<blocks_per_stream, TPB,0,streams[i]>>>(d_values_temp_i,d_csc,d_succ,d_max,node_size,NUMSTRM);
            if(!HandleCUDAError(hipStreamSynchronize(streams[i]))){
                std::cout<<"Error synchronizing device for Zero_Rows_Max_Idx at stream "<<i<<endl;
            }
            Zero_Cols_Max_Idx<<<blocks_per_stream, TPB,0,streams[i]>>>(d_values_temp_i,d_csc,d_succ,d_max,node_size,edge_size,NUMSTRM);
            if(!HandleCUDAError(hipStreamSynchronize(streams[i]))){
                std::cout<<"Error synchronizing device for Zero_Cols_Max_Idx at stream "<<i<<endl;
            }
        }
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hiprandDestroyGenerator(gen);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout<<"Time taken: "<<milliseconds<<endl;
    Save_Data(file,blocks_per_stream, milliseconds, damping_factor, threshold, epochs);
    if(!HandleCUDAError(hipFree(d_csc))){
        std::cout<<"Error freeing d_csc"<<endl;
    }
    if(!HandleCUDAError(hipFree(d_succ))){
        std::cout<<"Error freeing d_succ"<<endl;
    }
    if(!HandleCUDAError(hipFree(d_vec))){
        std::cout<<"Error freeing d_vec"<<endl;
    }
    if(!HandleCUDAError(hipFree(d_res))){
        std::cout<<"Error freeing d_res"<<endl;
    }
    if(!HandleCUDAError(hipFree(d_values))){
        std::cout<<"Error freeing d_values"<<endl;
    }
    unsigned int* h_rand_idx = new unsigned int[node_size];
    unsigned int* h_rand_score_idx = new unsigned int[node_size];
    thrust::fill(h_rand_idx, h_rand_idx+node_size, 0);
    // Take the sum of the vectors and then sort them
    for(int i = 1; i<NUMSTRM;i++){
        float* store_stream_res_i = store_stream_res + i*node_size;
        Float_VectAdd<<<blocks_per_stream, TPB>>>(store_stream_res, store_stream_res_i, node_size);
        if(!HandleCUDAError(hipDeviceSynchronize())){
            std::cout<<"Error synchronizing device for Float_VectAdd at stream "<<i<<endl;
        }
    }
    //Get the top k indexes
    float* h_store_stream_res_fin = new float[node_size];
    if(!HandleCUDAError(hipMemcpy(h_store_stream_res_fin, store_stream_res, sizeof(float)*node_size, hipMemcpyDeviceToHost))){
        std::cout<<"Error copying store_stream_res to host"<<endl;
    }
    thrust::sequence(h_rand_score_idx, h_rand_score_idx+node_size,0);
    thrust::sort_by_key(h_store_stream_res_fin, h_store_stream_res_fin+node_size, h_rand_score_idx, thrust::greater<float>());
    thrust::sequence(h_rand_idx, h_rand_idx+node_size,0);
    thrust::sort_by_key(count, count+node_size, h_rand_idx, thrust::greater<unsigned int>());
    for(int i = 0; i < K; i++){
        seed_set[i] = h_rand_score_idx[i];
        cout<<count[i]<<endl;
    }
    Export_Counts(COUNT_PATH, count,h_rand_idx, node_size);
    Export_Scores(SCORE_PATH, h_store_stream_res_fin, h_rand_score_idx, node_size);
    // delete[] h_rand_idx;

    for(int i = 0; i<NUMSTRM;i++){
        if(!HandleCUDAError(hipStreamDestroy(streams[i]))){
            std::cout<<"Error destroying stream number "<<i<<endl;
        }
    }
    if(!HandleCUDAError(hipFree(store_stream_res))){
        std::cout<<"Error freeing store_stream_res"<<endl;
    }
    if(!HandleCUDAError(hipFree(rand_numbers))){
        std::cout<<"Error freeing rand_numbers"<<endl;
    }
    if(!HandleCUDAError(hipFree(rand_vec_init))){
        std::cout<<"Error freeing rand_vec_init"<<endl;
    }
    if(!HandleCUDAError(hipFree(rand_init))){
        std::cout<<"Error freeing rand_init"<<endl;
    }
    if(!HandleCUDAError(hipDeviceReset())){
        std::cout<<"Error resetting device"<<endl;
    }
    delete[] h_rand_vec_init;
    delete[] l2_norm_rand_vec_init;
    delete[] l2_norm_d_res;
    delete[] sum;
    delete[] tol;
    delete[] values;

}

__host__ void  RIM_rand_Mart_BFS_v3(unsigned int* csc, unsigned int* succ, unsigned int node_size, unsigned int edge_size, unsigned int* seed_set, float threshold, string file){
    float damping_factor =.3;
    hipDeviceProp_t prop;
    int device;
    hipGetDevice(&device);  // Get the current device
    hipGetDeviceProperties(&prop, device);  // Get the properties of the device

    int maxActiveBlocksPerMultiprocessor = prop.maxThreadsPerMultiProcessor / TPB;
    int maxActiveBlocks = prop.multiProcessorCount * maxActiveBlocksPerMultiprocessor;
    int blocks_per_stream = maxActiveBlocks/NUMSTRM;
    blocks_per_stream = 40;

    printf("Max active blocks: %d\n", maxActiveBlocks);
    hipStream_t* streams = (hipStream_t*)malloc(sizeof(hipStream_t)*NUMSTRM);
    for(int i = 0; i < NUMSTRM; i++){
        if(!HandleCUDAError(hipStreamCreate(&streams[i]))){
            std::cout<<"Error creating stream number "<<i<<endl;
        }
    }
    unsigned int num_walker = node_size/(NUMSTRM)+1;
    unsigned int epochs;
    unsigned int* d_csc;
    unsigned int* d_succ;
    unsigned int* count = new unsigned int[node_size];
    float* d_vec; //we will use the seed set as the PR vector and then transfer the top k to the actual seed set
    float* d_res;
    float* d_track_val;
    float* vec = new float[NUMSTRM*node_size];
    float* values = new float[NUMSTRM*edge_size];
    float* res = new float[NUMSTRM*node_size]; 
    float* tol = new float[NUMSTRM];
    float* temp_sum = new float[NUMSTRM];
    float* sum = new float[NUMSTRM];
    float* l2_norm_d_res = new float[NUMSTRM];
    float* l2_norm_rand_vec_init = new float[NUMSTRM];
    float* d_penality;
    float* d_values_temp;
    float* d_diff;
    float* d_res_temp;
    unsigned int* max_index = new unsigned int[NUMSTRM]; 
    float* penalty_sum = new float[NUMSTRM];
    unsigned int* d_idx;
    thrust::fill(count, count+node_size, 0);
    thrust::fill(sum, sum+NUMSTRM, 0.0f);
    thrust::fill(tol,tol+NUMSTRM, 100.0f);
    thrust::fill(res, res+NUMSTRM*node_size, 0.0f);
    thrust::fill(vec, vec+NUMSTRM*node_size, 1.0f/node_size);
    thrust::fill(values, values+NUMSTRM*edge_size, 1.0f);
    thrust::fill(penalty_sum, penalty_sum+NUMSTRM, 0.0f);
    thrust::fill(max_index, max_index+NUMSTRM, 0);
    unsigned int* d_check;
    unsigned int* d_max;
    if(!HandleCUDAError(hipMalloc((void**)&d_csc, sizeof(unsigned int)*(node_size+1)))){
        std::cout<<"Error allocating memory for d_csc"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_succ, sizeof(unsigned int)*(edge_size)))){
        std::cout<<"Error allocating memory for d_succ"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_vec, sizeof(float)*node_size*NUMSTRM))){
        std::cout<<"Error allocating memory for d_seed_set"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_res, sizeof(float)*node_size*NUMSTRM))){
        std::cout<<"Error allocating memory for d_res"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_track_val, sizeof(float)*node_size*NUMSTRM))){
        std::cout<<"Error allocating memory for d_res"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_check, sizeof(unsigned int)*NUMSTRM*node_size))){
        std::cout<<"Error allocating memory for d_check"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_penality, sizeof(float)*node_size*NUMSTRM))){
        std::cout<<"Error allocating memory for d_penality"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_max, sizeof(unsigned int)*NUMSTRM))){
        std::cout<<"Error allocating memory for d_max"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_values_temp, sizeof(float)*edge_size*NUMSTRM))){
        std::cout<<"Error allocating memory for d_values_temp"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_diff, sizeof(float)*NUMSTRM*node_size))){
        std::cout<<"Error allocating memory for d_diff"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_res_temp, sizeof(float)*NUMSTRM*node_size))){
        std::cout<<"Error allocating memory for d_res_temp"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_idx, sizeof(unsigned int)*NUMSTRM*node_size))){
        std::cout<<"Error allocating memory for d_idx"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_csc, csc, sizeof(unsigned int)*(node_size+1), hipMemcpyHostToDevice))){
        std::cout<<"Error copying csc to device"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_succ, succ, sizeof(unsigned int)*edge_size, hipMemcpyHostToDevice))){
        std::cout<<"Error copying succ to device"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_vec, vec, sizeof(float)*node_size*NUMSTRM, hipMemcpyHostToDevice))){
        std::cout<<"Error copying vec to device"<<endl;
    }
    delete[] vec;
    if(!HandleCUDAError(hipMemcpy(d_res, res, sizeof(float)*node_size*NUMSTRM, hipMemcpyHostToDevice))){
        std::cout<<"Error copying res to device"<<endl;
    }
    delete[] res;

    
    float* d_values;
    if(!HandleCUDAError(hipMalloc((void**)&d_values, sizeof(float)*(edge_size*NUMSTRM)))){
        std::cout<<"Error allocating memory for d_values"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_values, values, sizeof(float)*edge_size*NUMSTRM, hipMemcpyHostToDevice))){
        std::cout<<"Error copying values to device"<<endl;
    }
    // delete[] values;


    float* rand_init;
    if(!HandleCUDAError(hipMalloc((void**)&rand_init, NUMSTRM*num_walker*sizeof(float)))){
        std::cout<<"Error allocating memory for rand_frog"<<endl;
    }
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);

    /*Now, we have the random numbers generated*/
    float* rand_vec_init;
    float* h_rand_vec_init = new float[node_size*NUMSTRM];
    thrust::fill(h_rand_vec_init, h_rand_vec_init+node_size*NUMSTRM, 0.0f);
    if(!HandleCUDAError(hipMalloc((void**)&rand_vec_init, sizeof(float)*node_size*NUMSTRM))){
        std::cout<<"Error allocating memory for rand_vec_init"<<endl;
    } 
    if(!HandleCUDAError(hipMemcpy(rand_vec_init, h_rand_vec_init, sizeof(float)*node_size*NUMSTRM, hipMemcpyHostToDevice))){
        std::cout<<"Error copying h_rand_vec_init to device"<<endl;
    }

    float* store_stream_res;
    float* h_store_stream_res = new float[node_size*NUMSTRM];
    if(!HandleCUDAError(hipMalloc((void**)&store_stream_res, sizeof(float)*node_size*NUMSTRM))){
        std::cout<<"Error allocating memory for store_stream_res"<<endl;
    }
    thrust::fill(thrust::device, store_stream_res, store_stream_res+node_size*NUMSTRM, 0.0f);
    delete[] h_store_stream_res;
    float* rand_numbers;
    if (!HandleCUDAError(hipMalloc((void**)&rand_numbers, sizeof(float) * edge_size*NUMSTRM))) {
        std::cout << "Error allocating memory for rand_numbers" << endl;
    }
    thrust::fill(thrust::device, d_penality, d_penality+node_size, 1.0f);
    printCudaMemoryUsage();
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    float tol_thresh=1;
    // epochs = (K)*(node_size/(NUMSTRM)+1);
    // epochs = 500*(K/NUMSTRM+1);
    epochs = (node_size/(600*NUMSTRM)+1)*(K/2);
    cout<<"Epochs: "<<epochs<<endl;
    for(int i = 0; i<NUMSTRM;i++){
        //Fill the values list prior to the start of the algorithm
        thrust::fill(thrust::device.on(streams[i]), d_values+i*edge_size, d_values+i*edge_size+edge_size, 1.0f);
        // thrust::fill(thrust::device.on(streams[i]), d_values_temp+i*edge_size, d_values_temp+i*edge_size+edge_size, 1.0f);
    }
    for(int k = 0; k < epochs; k++){
        // std::cout<<"Epoch "<<k<<endl;
        thrust::fill(tol,tol+NUMSTRM, 100.0f);
        thrust::fill(sum, sum+NUMSTRM, 0.0f);
        thrust::fill(temp_sum, temp_sum+NUMSTRM, 0.0f);
        srand(time(0));
        int rand_seed = rand();
        hiprandSetPseudoRandomGeneratorSeed(gen, rand_seed);
        hiprandGenerateUniform(gen, rand_init, num_walker*NUMSTRM);
        for(int i = 0; i < NUMSTRM; i++){
            //Initialize the random vector
            float* rand_init_i = rand_init + i*num_walker;
            float* rand_vec_init_i = rand_vec_init + i*node_size;
            unsigned int* d_check_i = d_check + i*node_size;
            float* d_diff_i = d_diff + i*node_size;
            float* d_res_temp_i = d_res_temp + i*node_size;
            thrust::fill(thrust::device.on(streams[i]),d_res+i*node_size, d_res+i*node_size+node_size, 0.0f);
            thrust::fill(thrust::device.on(streams[i]),d_track_val+i*node_size, d_track_val+i*node_size+node_size, 0.0f);
            thrust::fill(thrust::device.on(streams[i]),d_diff_i,d_diff_i+node_size,0.0f);
            thrust::fill(thrust::device.on(streams[i]),d_res_temp_i,d_res_temp_i+node_size,0.0f);
            Init_Random<<<blocks_per_stream, TPB,0,streams[i]>>>(rand_vec_init_i, rand_init_i, node_size, num_walker);
            if(!HandleCUDAError(hipStreamSynchronize(streams[i]))){
                std::cout<<"Error synchronizing device at Init Random for Stream "<<i<<endl;
            }
            thrust::copy(thrust::device.on(streams[i]),rand_vec_init_i,rand_vec_init_i+node_size, d_check_i);
        }
        int while_count = 0;
        bool check = true;
        while(check){
            while_count++;
            for(int i = 0; i < NUMSTRM; i++){
                //Perform the first iteration of the algorithm
                if(tol[i] > 0){
                    float* rand_vec_init_i = rand_vec_init + i*node_size;
                    float* d_res_i = d_res + i*node_size;
                    float* d_values_i = d_values + i*edge_size;
                    float* d_track_i = d_track_val + i*node_size;
                    unsigned int* d_check_i = d_check + i*node_size;
                    float* d_penality_i = d_penality + i*node_size;
                    float* d_diff_i = d_diff + i*node_size; 
                    float* d_res_temp_i = d_res_temp + i*node_size;
                    sparseCSRMat_Vec_Mult_Mart_BFS<unsigned int><<<blocks_per_stream, TPB,0,streams[i]>>>(d_csc, d_succ, d_values_i, rand_vec_init_i, d_res_i,d_check_i, threshold, node_size);  
                    if(!HandleCUDAError(hipStreamSynchronize(streams[i]))){
                        std::cout<<"Error synchronizing device at sparseCSRMat_Vec_Mult for stream "<<i<<endl;
                        exit(0);
                    }
                    thrust::copy(thrust::device.on(streams[i]), d_res_i, d_res_i+node_size, d_res_temp+i*node_size);
                    sum[i] = thrust::reduce(thrust::device.on(streams[i]), d_res_temp_i, d_res_temp_i+node_size);
                    float temp = sum[i];
                    thrust::transform(thrust::device.on(streams[i]), d_res_i, d_res_i+node_size, d_res_i, [=] __device__ (float x) { return x/temp; });
                    
                    thrust::transform(thrust::device.on(streams[i]), d_res_i, d_res_i+node_size, rand_vec_init_i, d_diff_i, thrust::minus<float>());
                    thrust::transform(thrust::device.on(streams[i]), d_diff_i, d_diff_i+node_size, d_diff_i, [] __device__ (float x) { return x * x; });
                    tol[i] = sum[i]-temp_sum[i];
                    temp_sum[i] = sum[i];
                    // cout<<"Tol: "<<tol[i]<<endl;    
                    thrust::copy(thrust::device.on(streams[i]), d_res_i, d_res_i+node_size, rand_vec_init_i);
                    thrust::fill(thrust::device.on(streams[i]), d_res_i, d_res_i+node_size, 0.0f);
                    thrust::fill(thrust::device.on(streams[i]), d_res_temp_i, d_res_temp_i+node_size, 0.0f);
                }
            }
            //In circumstance where we want all the be less than
            check = false;
            for(int i=0;i<NUMSTRM;i++){
                if(tol[i] > 1e-5){
                    check = true;
                }
            }
            //In circumstance we want one to be less than
            // check = true;
            // for(int i=0;i<NUMSTRM;i++){
            //     if(tol[i] <= 1e-5){
            //         check = false;
            //     }
            // }
        }
        for(int i = 0; i<NUMSTRM;i++){
            float* rand_vec_init_i = rand_vec_init + i*node_size;
            float* store_stream_res_i = store_stream_res + i*node_size;
            float* d_penality_i = d_penality + i*node_size;
            float* d_res_temp_i = d_res_temp + i*node_size;
            unsigned int* d_idx_i = d_idx + i*node_size;
            //Take the softmax of rand_vec_init_i
            thrust::copy(thrust::device.on(streams[i]), rand_vec_init_i, rand_vec_init_i+node_size, d_res_temp_i);
            sum[i] = thrust::reduce(thrust::device.on(streams[i]), d_res_temp_i, d_res_temp_i+node_size);
            float temp = sum[i];
            // cout<<"Sum: "<<sum[i]<<endl;
            if(sum[i]>0){
                thrust::sequence(thrust::device.on(streams[i]), d_idx_i, d_idx_i+node_size,0);
                thrust::transform(thrust::device.on(streams[i]), rand_vec_init_i, rand_vec_init_i+node_size, rand_vec_init_i, [=] __device__ (float x) { return x/temp; });
                //Find the index of the maximum element in the vector
                float* iter = thrust::max_element(thrust::device.on(streams[i]), rand_vec_init_i, rand_vec_init_i+node_size);
                max_index[i] = iter - rand_vec_init_i;
                count[max_index[i]]++;
                thrust::copy(thrust::device.on(streams[i]), rand_vec_init_i, rand_vec_init_i+node_size, d_penality_i);
                thrust::sort_by_key(thrust::device.on(streams[i]), d_penality_i, d_penality_i+node_size, d_idx_i, thrust::greater<float>());
                //Changes here
                Float_VectAdd_Cap<<<blocks_per_stream, TPB,0,streams[i]>>>(store_stream_res_i,rand_vec_init_i,d_idx_i, node_size, K);
                if(!HandleCUDAError(hipStreamSynchronize(streams[i]))){
                    std::cout<<"Error synchronizing device for Float_VectAdd at stream "<<i<<endl;
                }
                thrust::fill(thrust::device.on(streams[i]), rand_vec_init_i, rand_vec_init_i+node_size, 0);

            }
            unsigned int nmstrm = NUMSTRM;
            // thrust::transform(thrust::device.on(streams[i]), d_penality, d_penality+node_size, d_penality, [=] __device__ (float x) { return (x/nmstrm); });
        }
        if(!HandleCUDAError(hipMemcpy(d_max, max_index, sizeof(unsigned int)*NUMSTRM, hipMemcpyHostToDevice))){
            std::cout<<"Error copying max_index to device"<<endl;
        }
        for(int i=0; i<NUMSTRM;i++){
            float* d_values_temp_i = d_values + i*edge_size;
            Zero_Rows_Max_Idx<<<blocks_per_stream, TPB,0,streams[i]>>>(d_values_temp_i,d_csc,d_succ,d_max,node_size,NUMSTRM);
            if(!HandleCUDAError(hipStreamSynchronize(streams[i]))){
                std::cout<<"Error synchronizing device for Zero_Rows_Max_Idx at stream "<<i<<endl;
            }
            Zero_Cols_Max_Idx<<<blocks_per_stream, TPB,0,streams[i]>>>(d_values_temp_i,d_csc,d_succ,d_max,node_size,edge_size,NUMSTRM);
            if(!HandleCUDAError(hipStreamSynchronize(streams[i]))){
                std::cout<<"Error synchronizing device for Zero_Cols_Max_Idx at stream "<<i<<endl;
            }
        }
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hiprandDestroyGenerator(gen);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout<<"Time taken: "<<milliseconds<<endl;
    Save_Data(file,blocks_per_stream, milliseconds, damping_factor, threshold, epochs);
    if(!HandleCUDAError(hipFree(d_csc))){
        std::cout<<"Error freeing d_csc"<<endl;
    }
    if(!HandleCUDAError(hipFree(d_succ))){
        std::cout<<"Error freeing d_succ"<<endl;
    }
    if(!HandleCUDAError(hipFree(d_vec))){
        std::cout<<"Error freeing d_vec"<<endl;
    }
    if(!HandleCUDAError(hipFree(d_res))){
        std::cout<<"Error freeing d_res"<<endl;
    }
    if(!HandleCUDAError(hipFree(d_values))){
        std::cout<<"Error freeing d_values"<<endl;
    }
    unsigned int* h_rand_idx = new unsigned int[node_size];
    unsigned int* h_rand_score_idx = new unsigned int[node_size];
    thrust::fill(h_rand_idx, h_rand_idx+node_size, 0);
    // Take the sum of the vectors and then sort them
    float* d_store_res_fin;
    if(!HandleCUDAError(hipMalloc((void**)&d_store_res_fin, sizeof(float)*node_size))){
        std::cout<<"Error allocating memory for d_store_res_fin"<<endl;
    }
    thrust::fill(thrust::device, d_store_res_fin, d_store_res_fin+node_size, 0.0f);
    Condense_Score<<<blocks_per_stream, TPB>>>(d_store_res_fin, store_stream_res, node_size, NUMSTRM);
    if(!HandleCUDAError(hipDeviceSynchronize())){
        std::cout<<"Error synchronizing device for Condense_Score"<<endl;
    }
    //Get the top k indexes
    float* h_store_stream_res_fin = new float[node_size*NUMSTRM];
    if(!HandleCUDAError(hipMemcpy(h_store_stream_res_fin, d_store_res_fin, sizeof(float)*node_size, hipMemcpyDeviceToHost))){
        std::cout<<"Error copying store_stream_res to host"<<endl;
    }
    // thrust::sequence(h_rand_score_idx, h_rand_score_idx+node_size,0);
    // thrust::sort_by_key(h_store_stream_res_fin, h_store_stream_res_fin+node_size, h_rand_score_idx, thrust::greater<float>());
    thrust::sequence(h_rand_idx, h_rand_idx+node_size,0);
    ValueTuple* count_tuple = new ValueTuple[node_size];
    Make_Tuple_Count(count,h_rand_idx,count_tuple, node_size);
    thrust::sort_by_key(h_store_stream_res_fin,h_store_stream_res_fin+node_size,count_tuple, thrust::greater<float>());
    // thrust::default_random_engine g;
    // thrust::shuffle(thrust::host, count, count + node_size, g);
    //Need to find a way to shuffle the indexes that have the same values, shuffle based on score first, then by count
    Split_Tuple_Count(count, h_rand_idx, count_tuple, node_size);
    Export_Scores(SCORE_PATH, h_store_stream_res_fin, h_rand_idx, node_size);
    thrust::sort_by_key(count, count+node_size, h_rand_idx, thrust::greater<unsigned int>());

    for(int i = 0; i < K; i++){
        seed_set[i] = h_rand_idx[i];
        cout<<count[i]<<endl;
    }
    Export_Counts(COUNT_PATH, count,h_rand_idx, node_size);
    // delete[] h_rand_idx;

    for(int i = 0; i<NUMSTRM;i++){
        if(!HandleCUDAError(hipStreamDestroy(streams[i]))){
            std::cout<<"Error destroying stream number "<<i<<endl;
        }
    }
    if(!HandleCUDAError(hipFree(store_stream_res))){
        std::cout<<"Error freeing store_stream_res"<<endl;
    }
    if(!HandleCUDAError(hipFree(rand_numbers))){
        std::cout<<"Error freeing rand_numbers"<<endl;
    }
    if(!HandleCUDAError(hipFree(rand_vec_init))){
        std::cout<<"Error freeing rand_vec_init"<<endl;
    }
    if(!HandleCUDAError(hipFree(rand_init))){
        std::cout<<"Error freeing rand_init"<<endl;
    }
    if(!HandleCUDAError(hipDeviceReset())){
        std::cout<<"Error resetting device"<<endl;
    }
    delete[] h_rand_vec_init;
    delete[] l2_norm_rand_vec_init;
    delete[] l2_norm_d_res;
    delete[] sum;
    delete[] tol;
    delete[] values;

}

template <typename T>
__global__ void Copy(T* d_res, T* d_vec, unsigned int node_size){
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    for(int i = idx; i < node_size; i += blockDim.x * gridDim.x){
        d_vec[i] = d_res[i];
    }
}

__global__ void Transform_Bool(float* d_res, unsigned int* d_vec, unsigned int node_size){
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    for(int i = idx; i < node_size; i += blockDim.x * gridDim.x){
        if(abs(d_res[i]) > 0.0f){
            d_vec[i] = 1;
            // printf("d_res[i]: %f\n", d_res[i]);
        }
        else{
            d_vec[i] = 0;
            // printf("d_res[i]: %f\n", d_res[i]);
        }
    }
}

__global__ void Calc_Penalty(float* d_res, float* d_penality, unsigned int node_size){
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    for(int i = idx; i < node_size; i += blockDim.x * gridDim.x){
        d_penality[i] += 1-d_res[i];
    }
}

__global__ void Condense_Score(float* fin, float* inter, unsigned int node_size, unsigned int num_strm){
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    for(int i = idx; i < node_size; i += blockDim.x * gridDim.x){
        for(int j = 0; j < num_strm; j++){
            fin[i] += inter[i+j*node_size];
        }
    }
}