#include "hip/hip_runtime.h"
#include "../include/data.h"
/*The code for this come from Tang et al IMM Algorithm
The psuedo code is as follows:
1) Initialize a set R={empty set} and an integer LB = 1
2) Let epsilon'=sqrt(2)epsilon
3) for i = 1 to log_2(n)-1 do:
4) 	Let x=n/2^i
5) 	Let theta = lambda'/x
6)  while |R|<theta_i do:
7)      Select a node from G uniformly at random
8)      Generate an RR set for v, and insert it into R
9)  Let S_i = NodeSelection(R)
10) if n*FR(S)>=(1+epsilon')*x then:
11)     LB = n*FR(S)/(1+epsilon')
12)     break
13) Let theta= lambda^{*}/LB
14) while |R|<theta do:
15)     Select a node from G uniformly at random
16)     Generate an RR set for v, and insert it into R
17) Return R


What is going to be needed:
1) A graph
2) A set of RR sets
3) A set of nodes
4) A set of edges

How do we want to store these:
1) The RR sets would be traversable with CSC format, and any forward traversal will need
but would it be convenient to traverse both ways with a COO format?
2) We may be able to use linked lists as well but this will be slower, but also harder to implement, and harder for generating RRR sets

*/


__host__ void  RIM_rand_Ver1(unsigned int* csc, unsigned int* succ, unsigned int node_size, unsigned int edge_size, unsigned int* seed_set){
    float threshold = 0.75;
    hipDeviceProp_t prop;
    int device;
    hipGetDevice(&device);  // Get the current device
    hipGetDeviceProperties(&prop, device);  // Get the properties of the device

    int maxActiveBlocksPerMultiprocessor = prop.maxThreadsPerMultiProcessor / TPB;
    int maxActiveBlocks = prop.multiProcessorCount * maxActiveBlocksPerMultiprocessor;
    int blocks_per_stream = maxActiveBlocks/NUMSTRM;

    printf("Max active blocks: %d\n", maxActiveBlocks);
    hipStream_t* streams = (hipStream_t*)malloc(sizeof(hipStream_t)*NUMSTRM);
    for(int i = 0; i < NUMSTRM; i++){
        if(!HandleCUDAError(hipStreamCreate(&streams[i]))){
            cout<<"Error creating stream number "<<i<<endl;
        }
    }
    unsigned int* d_csc;
    unsigned int* d_succ;
    float* d_vec; //we will use the seed set as the PR vector and then transfer the top k to the actual seed set
    float* d_res;
    float* vec = new float[node_size];
    float* values = new float[edge_size];
    float* res = new float[node_size]; 
    float* tol = new float[NUMSTRM];
    float* sum = new float[NUMSTRM];
    float* l2_norm_d_vec = new float[NUMSTRM];
    float* l2_norm_rand_vec_init = new float[NUMSTRM];
    thrust::fill(sum, sum+NUMSTRM, 0.0f);
    thrust::fill(tol,tol+NUMSTRM, 100.0f);
    thrust::fill(res, res+node_size, 0.0f);
    thrust::fill(vec, vec+node_size, 1.0f/node_size);
    thrust::fill(values, values+edge_size, 1.0f);
    if(!HandleCUDAError(hipMalloc((void**)&d_csc, sizeof(unsigned int)*(node_size+1)))){
        cout<<"Error allocating memory for d_csc"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_succ, sizeof(unsigned int)*(edge_size)))){
        cout<<"Error allocating memory for d_succ"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_vec, sizeof(float)*node_size))){
        cout<<"Error allocating memory for d_seed_set"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_res, sizeof(float)*node_size))){
        cout<<"Error allocating memory for d_res"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_csc, csc, sizeof(unsigned int)*node_size, hipMemcpyHostToDevice))){
        cout<<"Error copying csc to device"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_succ, succ, sizeof(unsigned int)*edge_size, hipMemcpyHostToDevice))){
        cout<<"Error copying succ to device"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_vec, vec, sizeof(float)*node_size, hipMemcpyHostToDevice))){
        cout<<"Error copying vec to device"<<endl;
    }
    delete[] vec;
    if(!HandleCUDAError(hipMemcpy(d_res, res, sizeof(float)*node_size, hipMemcpyHostToDevice))){
        cout<<"Error copying res to device"<<endl;
    }
    delete[] res;
    
    float* d_values;
    if(!HandleCUDAError(hipMalloc((void**)&d_values, sizeof(float)*(edge_size)))){
        cout<<"Error allocating memory for d_values"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_values, values, sizeof(float)*edge_size, hipMemcpyHostToDevice))){
        cout<<"Error copying values to device"<<endl;
    }
    // delete[] values;
    if(!HandleCUDAError(hipMemcpy(values,d_values, sizeof(float)*edge_size, hipMemcpyDeviceToHost))){
        cout<<"Error copying values to device"<<endl;
    }

    // unsigned int num_blocks = (node_size+TPB-1)/TPB;
    // unsigned int num_blocks2 = (edge_size+TPB-1)/TPB;
    float* rand_init;
    if(!HandleCUDAError(hipMalloc((void**)&rand_init, K*sizeof(float)))){
        std::cout<<"Error allocating memory for rand_frog"<<endl;
    }
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    srand(time(0));
    int rand_seed = rand();
    hiprandSetPseudoRandomGeneratorSeed(gen, rand_seed);
    hiprandGenerateUniform(gen, rand_init, K);
    /*Now, we have the random numbers generated*/
    hiprandDestroyGenerator(gen);
    float* rand_vec_init;
    float* h_rand_vec_init = new float[node_size]{0.0f};
    if(!HandleCUDAError(hipMalloc((void**)&rand_vec_init, sizeof(float)*node_size))){
        std::cout<<"Error allocating memory for rand_vec_init"<<endl;
    } 
    if(!HandleCUDAError(hipMemcpy(rand_vec_init, h_rand_vec_init, sizeof(float)*node_size, hipMemcpyHostToDevice))){
        cout<<"Error copying h_rand_vec_init to device"<<endl;
    }
    while(tol[0] > threshold && tol[1] > threshold && tol[2] > threshold && tol[3] > threshold && tol[4] > threshold && tol[5] > threshold){
        for(int i = 0; i < NUMSTRM; i++){
            //Initialize the random vector
            if(tol[i] > threshold){
                Init_Random<<<blocks_per_stream, TPB,0,streams[i]>>>(rand_vec_init, rand_init, node_size, K);
                if(!HandleCUDAError(hipStreamSynchronize(streams[i]))){
                    cout<<"Error synchronizing device at Init Random for Stream "<<i<<endl;
                }
            }
        }
        for(int i = 0; i < NUMSTRM; i++){
            //Perform the first iteration of the algorithm
            if(tol[i] > threshold){
                sparseCSRMat_Vec_Mult<<<blocks_per_stream, TPB,0,streams[i]>>>(d_csc, d_succ, d_values, rand_vec_init, d_res, node_size);  
                if(!HandleCUDAError(hipStreamSynchronize(streams[i]))){
                    cout<<"Error synchronizing device at sparseCSRMat_Vec_Mult for stream "<<i<<endl;
                }
            }
        }
            // Just Verify that the multiplication is working
        for(int i = 0; i < NUMSTRM; i++){
            // Add 1/n to the vector
            if(tol[i] > threshold){
                Float_VectAdd<<<blocks_per_stream, TPB,0,streams[i]>>>(d_vec, rand_vec_init, node_size);
                if(!HandleCUDAError(hipStreamSynchronize(streams[i]))){
                    cout<<"Error synchronizing device for Float_VectAdd at stream "<<i<<endl;
                }
                if(!HandleCUDAError(hipMemcpy(rand_vec_init, d_vec, sizeof(float)*node_size, hipMemcpyDeviceToDevice))){
                    cout<<"Error copying d_vec to host"<<endl;
                }
                //Need to normalize the vector using thrust library

                l2_norm_d_vec[i] = thrust::transform_reduce(thrust::device, d_vec, d_vec + node_size, [] __device__ (float x) { return x * x; }, 0.0f, thrust::plus<float>());
                l2_norm_d_vec[i] = sqrt(l2_norm_d_vec[i]);

                l2_norm_rand_vec_init[i] = thrust::transform_reduce(thrust::device, rand_vec_init, rand_vec_init + node_size, [] __device__ (float x) { return x * x; }, 0.0f, thrust::plus<float>());
                l2_norm_rand_vec_init[i] = sqrt(l2_norm_rand_vec_init[i]);


                sum[i] = thrust::reduce(thrust::device.on(streams[i]), rand_vec_init, rand_vec_init+node_size);
                thrust::transform(thrust::device.on(streams[i]), rand_vec_init, rand_vec_init+node_size, rand_vec_init, thrust::placeholders::_1/sum);
                thrust::fill(thrust::device.on(streams[i]), d_vec, d_vec+node_size, 1.0f/node_size);
                if(!HandleCUDAError(hipStreamSynchronize(streams[i]))){
                    cout<<"Error synchronizing device"<<endl;
                }
            }
        }
    }
}



__host__ void CheckSparseMatVec(unsigned int* csc, unsigned int* succ,edge* edge_list, unsigned int node_size, unsigned int edge_size){
    unsigned int* d_csc;
    unsigned int* d_succ;
    float* d_vec; //we will use the seed set as the PR vector and then transfer the top k to the actual seed set
    float* d_res;
    float* vec = new float[node_size];
    float* values = new float[edge_size];
    float* res = new float[node_size]; 
    thrust::fill(res, res+node_size, 0.0f);
    thrust::fill(vec, vec+node_size, 1.0f/node_size);
    thrust::fill(values, values+edge_size, 1.0f);
    hipDeviceProp_t prop;
    int device;
    hipGetDevice(&device);  // Get the current device
    hipGetDeviceProperties(&prop, device);  // Get the properties of the device

    int maxActiveBlocksPerMultiprocessor = prop.maxThreadsPerMultiProcessor / TPB;
    int maxActiveBlocks = prop.multiProcessorCount * maxActiveBlocksPerMultiprocessor;
    int blocks_per_stream = maxActiveBlocks/NUMSTRM;

    printf("Max active blocks: %d\n", maxActiveBlocks);
    hipStream_t* streams = (hipStream_t*)malloc(sizeof(hipStream_t)*NUMSTRM);
    for(int i = 0; i < NUMSTRM; i++){
        if(!HandleCUDAError(hipStreamCreate(&streams[i]))){
            cout<<"Error creating stream number "<<i<<endl;
        }
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_csc, sizeof(unsigned int)*(node_size+1)))){
        cout<<"Error allocating memory for d_csc"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_succ, sizeof(unsigned int)*(edge_size)))){
        cout<<"Error allocating memory for d_succ"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_vec, sizeof(float)*node_size))){
        cout<<"Error allocating memory for d_seed_set"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_res, sizeof(float)*node_size))){
        cout<<"Error allocating memory for d_res"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_csc, csc, sizeof(unsigned int)*node_size, hipMemcpyHostToDevice))){
        cout<<"Error copying csc to device"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_succ, succ, sizeof(unsigned int)*edge_size, hipMemcpyHostToDevice))){
        cout<<"Error copying succ to device"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_vec, vec, sizeof(float)*node_size, hipMemcpyHostToDevice))){
        cout<<"Error copying vec to device"<<endl;
    }
    delete[] vec;
    if(!HandleCUDAError(hipMemcpy(d_res, res, sizeof(float)*node_size, hipMemcpyHostToDevice))){
        cout<<"Error copying res to device"<<endl;
    }
    delete[] res;
    
    float* d_values;
    if(!HandleCUDAError(hipMalloc((void**)&d_values, sizeof(float)*(edge_size)))){
        cout<<"Error allocating memory for d_values"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_values, values, sizeof(float)*edge_size, hipMemcpyHostToDevice))){
        cout<<"Error copying values to device"<<endl;
    }
    // delete[] values;
    // if(!HandleCUDAError(hipMemcpy(values,d_values, sizeof(float)*edge_size, hipMemcpyDeviceToHost))){
    //     cout<<"Error copying values to device"<<endl;
    // }

    float* rand_init;
    if(!HandleCUDAError(hipMalloc((void**)&rand_init, K*sizeof(float)))){
        std::cout<<"Error allocating memory for rand_frog"<<endl;
    }
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    srand(time(0));
    int rand_seed = rand();
    hiprandSetPseudoRandomGeneratorSeed(gen, rand_seed);
    hiprandGenerateUniform(gen, rand_init, K);
    /*Now, we have the random numbers generated*/
    hiprandDestroyGenerator(gen);
    float* rand_vec_init;
    float* h_rand_vec_init = new float[node_size]{0.0f};
    if(!HandleCUDAError(hipMalloc((void**)&rand_vec_init, sizeof(float)*node_size))){
        std::cout<<"Error allocating memory for rand_vec_init"<<endl;
    } 
    if(!HandleCUDAError(hipMemcpy(rand_vec_init, h_rand_vec_init, sizeof(float)*node_size, hipMemcpyHostToDevice))){
        cout<<"Error copying h_rand_vec_init to device"<<endl;
    }
    //Initialize the random vector
    Init_Random<<<blocks_per_stream, TPB>>>(rand_vec_init, rand_init, node_size, K);
    if(!HandleCUDAError(hipDeviceSynchronize())){
        cout<<"Error synchronizing device"<<endl;
    }
    //Perform the first iteration of the algorithm
    sparseCSRMat_Vec_Mult<<<blocks_per_stream, TPB>>>(d_csc, d_succ, d_values, rand_vec_init, d_res, node_size);  
    if(!HandleCUDAError(hipDeviceSynchronize())){
        cout<<"Error synchronizing device"<<endl;
    }
    Float_VectAdd<<<blocks_per_stream, TPB>>>(d_res, d_vec, node_size);
    if(!HandleCUDAError(hipDeviceSynchronize())){
        cout<<"Error synchronizing device"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(h_rand_vec_init, rand_vec_init, sizeof(float)*node_size, hipMemcpyDeviceToHost))){
        cout<<"Error copying d_vec to host"<<endl;
    }
    float* h_res_GPU = new float[node_size]{0.0f};

    if(!HandleCUDAError(hipMemcpy(h_res_GPU, d_res, sizeof(float)*node_size, hipMemcpyDeviceToHost))){
        cout<<"Error copying d_vec to host"<<endl;
    }

    float* h_res_CPU = new float[node_size]{0.0f};
    float* A = (float*)malloc(sizeof(float)*node_size*node_size);
    GenAdj(edge_list, A, node_size, edge_size);
    h_MatVecMult(A, h_rand_vec_init, h_res_CPU, node_size);
    float* support_vec = new float[node_size];
    thrust::fill(support_vec, support_vec+node_size, 1.0f/node_size);
    for(int i = 0; i < node_size; i++){
        h_res_CPU[i] += support_vec[i];
    }
    float sum = 0.0f;
    sum = thrust::inner_product(thrust::device, d_res, d_res+node_size, d_res, 0.0f);
    sum = sqrt(sum);
    thrust::transform(thrust::device, d_res, d_res+node_size, d_res, thrust::placeholders::_1/sum);
    if(!HandleCUDAError(hipDeviceSynchronize())){
        cout<<"Error synchronizing device"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(h_res_GPU, d_res, sizeof(float)*node_size, hipMemcpyDeviceToHost))){
        cout<<"Error copying d_vec to host"<<endl;
    }
    Normalize_L2(h_res_CPU, node_size);
    Verify(h_res_GPU, h_res_CPU, node_size);

}

__global__ void sparseCSRMat_Vec_Mult(unsigned int* csc, unsigned int* succ, float* values, float* vec, float* result, unsigned int node_size){
    unsigned int tid = threadIdx.x + blockIdx.x*blockDim.x;
    for(int t = tid; t < node_size; t+=blockDim.x*gridDim.x){
        unsigned int start = csc[t];
        unsigned int end = csc[t+1];
        float sum = 0.0f;
        for(int i = start; i < end; i++){
            sum += values[i]*vec[succ[i]];
        }
        result[t] = sum;
    }
}

__global__ void Float_VectAdd(float* vec1, float* vec2, unsigned int size){
    unsigned int tid = threadIdx.x + blockIdx.x*blockDim.x;
    if(tid < size){
        vec1[tid] += vec2[tid];
    }
}

__global__ void Init_Random(float* vec, float* rand_init, unsigned int size, unsigned int k){
    unsigned int tid = threadIdx.x + blockIdx.x*blockDim.x;
    unsigned int idx =0;
    if(tid<k){
        rand_init[tid] = floorf(rand_init[tid]*size);
        idx = (unsigned int)rand_init[tid];
        idx = idx%size;
        vec[idx] = 1.0f;
    }
}

__host__ void Verify(float* gpu_vec, float* cpu_vec, unsigned int size){
    float sum = 0.0f;
    for(int i = 0; i < size; i++){
        sum = abs(gpu_vec[i]-cpu_vec[i]);
        if (sum > 0.00001f){
            cout<<"Error at index "<<i<<endl;
            cout<<"GPU: "<<gpu_vec[i]<<endl;
            cout<<"CPU: "<<cpu_vec[i]<<endl;
            return;
        }
    }
    cout<<"No errors found"<<endl;
}