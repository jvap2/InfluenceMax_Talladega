#include "hip/hip_runtime.h"
#include "../include/data.h"
/*The code for this come from Tang et al IMM Algorithm
The psuedo code is as follows:
1) Initialize a set R={empty set} and an integer LB = 1
2) Let epsilon'=sqrt(2)epsilon
3) for i = 1 to log_2(n)-1 do:
4) 	Let x=n/2^i
5) 	Let theta = lambda'/x
6)  while |R|<theta_i do:
7)      Select a node from G uniformly at random
8)      Generate an RR set for v, and insert it into R
9)  Let S_i = NodeSelection(R)
10) if n*FR(S)>=(1+epsilon')*x then:
11)     LB = n*FR(S)/(1+epsilon')
12)     break
13) Let theta= lambda^{*}/LB
14) while |R|<theta do:
15)     Select a node from G uniformly at random
16)     Generate an RR set for v, and insert it into R
17) Return R


What is going to be needed:
1) A graph
2) A set of RR sets
3) A set of nodes
4) A set of edges

How do we want to store these:
1) The RR sets would be traversable with CSC format, and any forward traversal will need
but would it be convenient to traverse both ways with a COO format?
2) We may be able to use linked lists as well but this will be slower, but also harder to implement, and harder for generating RRR sets

*/


__host__ void  RIM_rand_Ver1(unsigned int* csc, unsigned int* succ, unsigned int node_size, unsigned int edge_size, unsigned int* seed_set){
    float threshold = 0.75;
    hipStream_t* streams = (hipStream_t*)malloc(sizeof(hipStream_t)*NUMSTRM);
    for(int i = 0; i < NUMSTRM; i++){
        if(!HandleCUDAError(hipStreamCreate(&streams[i]))){
            cout<<"Error creating stream number "<<i<<endl;
        }
    }
    unsigned int* d_csc;
    unsigned int* d_succ;
    unsigned int* d_vec; //we will use the seed set as the PR vector and then transfer the top k to the actual seed set
    if(!HandleCUDAError(hipMalloc((void**)&d_csc, sizeof(unsigned int)*node_size))){
        cout<<"Error allocating memory for d_csc"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_succ, sizeof(unsigned int)*edge_size))){
        cout<<"Error allocating memory for d_succ"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_vec, sizeof(unsigned int)*node_size))){
        cout<<"Error allocating memory for d_seed_set"<<endl;
    }

    
}


__global__ void sparseCSRMat_Vec_Mult(unsigned int* csc, unsigned int* succ, unsigned int* vec, unsigned int* result, unsigned int node_size){
    unsigned int tid = threadIdx.x + blockIdx.x*blockDim.x;
    if(tid < node_size){
        unsigned int start = csc[tid];
        unsigned int end = csc[tid+1];
        unsigned int sum = 0;
        for(int i = start; i < end; i++){
            sum += vec[succ[i]];
        }
        result[tid] = sum;
    }
}