#include "hip/hip_runtime.h"
#include "../include/data.h"
/*The code for this come from Tang et al IMM Algorithm
The psuedo code is as follows:
1) Initialize a set R={empty set} and an integer LB = 1
2) Let epsilon'=sqrt(2)epsilon
3) for i = 1 to log_2(n)-1 do:
4) 	Let x=n/2^i
5) 	Let theta = lambda'/x
6)  while |R|<theta_i do:
7)      Select a node from G uniformly at random
8)      Generate an RR set for v, and insert it into R
9)  Let S_i = NodeSelection(R)
10) if n*FR(S)>=(1+epsilon')*x then:
11)     LB = n*FR(S)/(1+epsilon')
12)     break
13) Let theta= lambda^{*}/LB
14) while |R|<theta do:
15)     Select a node from G uniformly at random
16)     Generate an RR set for v, and insert it into R
17) Return R


What is going to be needed:
1) A graph
2) A set of RR sets
3) A set of nodes
4) A set of edges

How do we want to store these:
1) The RR sets would be traversable with CSC format, and any forward traversal will need
but would it be convenient to traverse both ways with a COO format?
2) We may be able to use linked lists as well but this will be slower, but also harder to implement, and harder for generating RRR sets

*/

__device__ float eval_values(float rand_num, float val,float threshold){
    if(rand_num > threshold){
        return val;
    }
    else{
        return 0.0f;
    }
}


__host__ void Save_Data(string file, float time, float damping_factor, float threshold,unsigned int epoch){
    ofstream myfile;
    myfile.open(file, std::ios_base::app);
    myfile<<NUMSTRM<<","<<time<<","<<damping_factor<<","<<threshold<<","<<epoch<<","<<K;
    myfile.close();
}

__host__ void  RIM_rand_Ver1(unsigned int* csc, unsigned int* succ, unsigned int node_size, unsigned int edge_size, unsigned int* seed_set, string file){
    float threshold = .2;
    float damping_factor =.3;
    hipDeviceProp_t prop;
    int device;
    hipGetDevice(&device);  // Get the current device
    hipGetDeviceProperties(&prop, device);  // Get the properties of the device

    int maxActiveBlocksPerMultiprocessor = prop.maxThreadsPerMultiProcessor / TPB;
    int maxActiveBlocks = prop.multiProcessorCount * maxActiveBlocksPerMultiprocessor;
    int blocks_per_stream = maxActiveBlocks/NUMSTRM+1;

    printf("Max active blocks: %d\n", maxActiveBlocks);
    hipStream_t* streams = (hipStream_t*)malloc(sizeof(hipStream_t)*NUMSTRM);
    for(int i = 0; i < NUMSTRM; i++){
        if(!HandleCUDAError(hipStreamCreate(&streams[i]))){
            cout<<"Error creating stream number "<<i<<endl;
        }
    }
    unsigned int num_walker = node_size/20;
    unsigned int epochs=30;
    unsigned int* d_csc;
    unsigned int* d_succ;
    float* d_vec; //we will use the seed set as the PR vector and then transfer the top k to the actual seed set
    float* d_res;
    float* vec = new float[NUMSTRM*node_size];
    float* values = new float[NUMSTRM*edge_size];
    float* res = new float[NUMSTRM*node_size]; 
    float* tol = new float[NUMSTRM];
    float* sum = new float[NUMSTRM];
    float* l2_norm_d_res = new float[NUMSTRM];
    float* l2_norm_rand_vec_init = new float[NUMSTRM];
    thrust::fill(sum, sum+NUMSTRM, 0.0f);
    thrust::fill(tol,tol+NUMSTRM, 100.0f);
    thrust::fill(res, res+NUMSTRM*node_size, 0.0f);
    thrust::fill(vec, vec+NUMSTRM*node_size, 1.0f/node_size);
    thrust::fill(values, values+NUMSTRM*edge_size, 1.0f);
    if(!HandleCUDAError(hipMalloc((void**)&d_csc, sizeof(unsigned int)*(node_size+1)))){
        cout<<"Error allocating memory for d_csc"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_succ, sizeof(unsigned int)*(edge_size)))){
        cout<<"Error allocating memory for d_succ"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_vec, sizeof(float)*node_size*NUMSTRM))){
        cout<<"Error allocating memory for d_seed_set"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_res, sizeof(float)*node_size*NUMSTRM))){
        cout<<"Error allocating memory for d_res"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_csc, csc, sizeof(unsigned int)*node_size, hipMemcpyHostToDevice))){
        cout<<"Error copying csc to device"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_succ, succ, sizeof(unsigned int)*edge_size, hipMemcpyHostToDevice))){
        cout<<"Error copying succ to device"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_vec, vec, sizeof(float)*node_size*NUMSTRM, hipMemcpyHostToDevice))){
        cout<<"Error copying vec to device"<<endl;
    }
    delete[] vec;
    if(!HandleCUDAError(hipMemcpy(d_res, res, sizeof(float)*node_size*NUMSTRM, hipMemcpyHostToDevice))){
        cout<<"Error copying res to device"<<endl;
    }
    delete[] res;

    
    float* d_values;
    if(!HandleCUDAError(hipMalloc((void**)&d_values, sizeof(float)*(edge_size*NUMSTRM)))){
        cout<<"Error allocating memory for d_values"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_values, values, sizeof(float)*edge_size*NUMSTRM, hipMemcpyHostToDevice))){
        cout<<"Error copying values to device"<<endl;
    }
    // delete[] values;


    // // unsigned int num_blocks = (node_size+TPB-1)/TPB;
    // // unsigned int num_blocks2 = (edge_size+TPB-1)/TPB;
    float* rand_init;
    if(!HandleCUDAError(hipMalloc((void**)&rand_init, NUMSTRM*num_walker*sizeof(float)))){
        std::cout<<"Error allocating memory for rand_frog"<<endl;
    }
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    srand(time(0));
    int rand_seed = rand();
    hiprandSetPseudoRandomGeneratorSeed(gen, rand_seed);
    hiprandGenerateUniform(gen, rand_init, num_walker*NUMSTRM);
    /*Now, we have the random numbers generated*/
    hiprandDestroyGenerator(gen);
    float* rand_vec_init;
    float* h_rand_vec_init = new float[node_size*NUMSTRM];
    thrust::fill(h_rand_vec_init, h_rand_vec_init+node_size*NUMSTRM, 0.0f);
    if(!HandleCUDAError(hipMalloc((void**)&rand_vec_init, sizeof(float)*node_size*NUMSTRM))){
        std::cout<<"Error allocating memory for rand_vec_init"<<endl;
    } 
    if(!HandleCUDAError(hipMemcpy(rand_vec_init, h_rand_vec_init, sizeof(float)*node_size*NUMSTRM, hipMemcpyHostToDevice))){
        cout<<"Error copying h_rand_vec_init to device"<<endl;
    }
    float* rand_numbers;
    if (!HandleCUDAError(hipMalloc((void**)&rand_numbers, sizeof(float) * edge_size*NUMSTRM))) {
        cout << "Error allocating memory for rand_numbers" << endl;
    }
    printCudaMemoryUsage();
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    for(int i = 0; i < NUMSTRM; i++){
        //Initialize the random vector
        float* rand_init_i = rand_init + i*num_walker;
        float* rand_vec_init_i = rand_vec_init + i*node_size;
        Init_Random<<<blocks_per_stream, TPB,0,streams[i]>>>(rand_vec_init_i, rand_init_i, node_size, num_walker);
        if(!HandleCUDAError(hipStreamSynchronize(streams[i]))){
            cout<<"Error synchronizing device at Init Random for Stream "<<i<<endl;
        }
    }
    for(int i = 0; i < epochs; i++){
        cout<<"Epoch "<<i<<endl;
        thrust::fill(tol,tol+NUMSTRM, 100.0f);
        int while_count = 0;
        while_count=0;
        while(thrust::all_of(thrust::host, tol, tol+NUMSTRM, [=] __device__ (float x) { return x > threshold; }) && while_count < 1000){
            while_count++;
            for(int i = 0; i < NUMSTRM; i++){
                //Perform the first iteration of the algorithm
                if(tol[i] > threshold){
                    float* rand_numbers_i = rand_numbers + i*NUMSTRM;
                    hiprandGenerator_t gen;
                    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
                    srand(time(0));
                    int rand_seed = rand();
                    hiprandSetPseudoRandomGeneratorSeed(gen, rand_seed);
                    hiprandGenerateUniform(gen, rand_numbers_i, edge_size);
                    hiprandDestroyGenerator(gen);
                    float* rand_vec_init_i = rand_vec_init + i*node_size;
                    float* d_res_i = d_res + i*node_size;
                    float* d_values_i = d_values + i*edge_size;
                    thrust::transform(thrust::device.on(streams[i]), rand_numbers_i, rand_numbers_i+edge_size, d_values_i, d_values_i, [threshold] __device__ (float x, float y) { return eval_values(x,y,threshold); });
                    sparseCSRMat_Vec_Mult<<<blocks_per_stream, TPB,0,streams[i]>>>(d_csc, d_succ, d_values_i, rand_vec_init_i, d_res_i, node_size);  
                    if(!HandleCUDAError(hipStreamSynchronize(streams[i]))){
                        cout<<"Error synchronizing device at sparseCSRMat_Vec_Mult for stream "<<i<<endl;
                    }
                    thrust::fill(thrust::device.on(streams[i]), d_values_i, d_values_i+edge_size, 1.0f);
                }
            }
            for(int i = 0; i < NUMSTRM; i++){
                // Add 1/n to the vector
                if(tol[i] > threshold){
                    float* d_res_i = d_res + i*node_size;
                    float* d_vec_i = d_vec + i*node_size;
                    float* rand_vec_init_i = rand_vec_init + i*node_size;
                    Float_VectAdd<<<blocks_per_stream, TPB,0,streams[i]>>>(d_res_i,d_vec_i, node_size);
                    if(!HandleCUDAError(hipStreamSynchronize(streams[i]))){
                        cout<<"Error synchronizing device for Float_VectAdd at stream "<<i<<endl;
                    }
                    //Need to normalize the vector using thrust library

                    l2_norm_d_res[i] = thrust::transform_reduce(thrust::device, d_res_i, d_res_i + node_size, [] __device__ (float x) { return x * x; }, 0.0f, thrust::plus<float>());
                    l2_norm_d_res[i] = sqrt(l2_norm_d_res[i]);

                    l2_norm_rand_vec_init[i] = thrust::transform_reduce(thrust::device, rand_vec_init_i, rand_vec_init_i + node_size, [] __device__ (float x) { return x * x; }, 0.0f, thrust::plus<float>());
                    l2_norm_rand_vec_init[i] = sqrt(l2_norm_rand_vec_init[i]);

                    tol[i] = abs(l2_norm_d_res[i]-l2_norm_rand_vec_init[i]);
                    thrust::copy(thrust::device.on(streams[i]), d_res_i, d_res_i+node_size, rand_vec_init_i);


                    sum[i] = thrust::reduce(thrust::device.on(streams[i]), rand_vec_init_i, rand_vec_init_i+node_size);
                    float temp = sum[i];
                    thrust::transform(thrust::device.on(streams[i]), rand_vec_init_i, rand_vec_init_i+node_size, rand_vec_init_i, [=] __device__ (float x) { return x/temp; });
                    // thrust::fill(thrust::device.on(streams[i]), d_vec, d_vec+node_size, 1.0f/node_size);
                }
            }
        }
        // switch(epochs%NUMSTRM){
        //     case 0:
        //         thrust::transform(thrust::device, rand_vec_init+(NUMSTRM-1)*edge_size, rand_vec_init+(NUMSTRM)*edge_size, rand_vec_init, [damping_factor] __device__ (float x) { return x + damping_factor * x; });
        //         break;
        //     default:
        //         thrust::transform(thrust::device, rand_vec_init+(epochs%NUMSTRM)*edge_size, rand_vec_init+(1+epochs%NUMSTRM)*edge_size+edge_size, rand_vec_init+(1+epochs%NUMSTRM)*edge_size, [damping_factor] __device__ (float x) { return x + damping_factor * x; });
        //         break;
            
        // }
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    cout<<"Time taken: "<<milliseconds<<endl;
    Save_Data(file, milliseconds, damping_factor, threshold, epochs);
    if(!HandleCUDAError(hipFree(d_csc))){
        cout<<"Error freeing d_csc"<<endl;
    }
    if(!HandleCUDAError(hipFree(d_succ))){
        cout<<"Error freeing d_succ"<<endl;
    }
    if(!HandleCUDAError(hipFree(d_vec))){
        cout<<"Error freeing d_vec"<<endl;
    }
    if(!HandleCUDAError(hipFree(d_res))){
        cout<<"Error freeing d_res"<<endl;
    }
    if(!HandleCUDAError(hipFree(d_values))){
        cout<<"Error freeing d_values"<<endl;
    }
    unsigned int* rand_idx;
    unsigned int* h_rand_idx = new unsigned int[node_size];
    thrust::fill(h_rand_idx, h_rand_idx+node_size, 0);
    if(!HandleCUDAError(hipMalloc((void**)&rand_idx, sizeof(unsigned int)*node_size))){
        cout<<"Error allocating memory for rand_idx"<<endl;
    }
    thrust::sequence(thrust::device, rand_idx, rand_idx+node_size);
    //Take the sum of the vectors and then sort them
    for(int i = 1; i<NUMSTRM;i++){
        thrust::transform(thrust::device, rand_vec_init, rand_vec_init+node_size, rand_vec_init+i*node_size, rand_vec_init, thrust::plus<float>());
    }
    thrust::sort_by_key(thrust::device, rand_vec_init, rand_vec_init+node_size, rand_idx, thrust::greater<float>());
    //Get the top k indexes
    if(!HandleCUDAError(hipMemcpy(h_rand_idx, rand_idx, sizeof(unsigned int)*K, hipMemcpyDeviceToHost))){
        cout<<"Error copying rand_idx to host"<<endl;
    }
    for(int i = 0; i < K; i++){
        seed_set[i] = h_rand_idx[i];
    }
    if(!HandleCUDAError(hipFree(rand_idx))){
        cout<<"Error freeing rand_idx"<<endl;
    }
    delete[] h_rand_idx;

    for(int i = 0; i<NUMSTRM;i++){
        if(!HandleCUDAError(hipStreamDestroy(streams[i]))){
            cout<<"Error destroying stream number "<<i<<endl;
        }
    }
}


__host__ void  RIM_rand_Ver2(unsigned int* csc, unsigned int* succ, unsigned int node_size, unsigned int edge_size, unsigned int* seed_set, string file){
    float threshold = .8;
    float damping_factor =.3;
    hipDeviceProp_t prop;
    int device;
    hipGetDevice(&device);  // Get the current device
    hipGetDeviceProperties(&prop, device);  // Get the properties of the device

    int maxActiveBlocksPerMultiprocessor = prop.maxThreadsPerMultiProcessor / TPB;
    int maxActiveBlocks = prop.multiProcessorCount * maxActiveBlocksPerMultiprocessor;
    int blocks_per_stream = maxActiveBlocks/NUMSTRM+1;

    printf("Max active blocks: %d\n", maxActiveBlocks);
    hipStream_t* streams = (hipStream_t*)malloc(sizeof(hipStream_t)*NUMSTRM);
    for(int i = 0; i < NUMSTRM; i++){
        if(!HandleCUDAError(hipStreamCreate(&streams[i]))){
            cout<<"Error creating stream number "<<i<<endl;
        }
    }
    unsigned int num_walker = node_size/20;
    unsigned int epochs=30;
    unsigned int* d_csc;
    unsigned int* d_succ;
    float* d_vec; //we will use the seed set as the PR vector and then transfer the top k to the actual seed set
    float* d_res;
    float* vec = new float[NUMSTRM*node_size];
    float* values = new float[NUMSTRM*edge_size];
    float* res = new float[NUMSTRM*node_size]; 
    float* tol = new float[NUMSTRM];
    float* sum = new float[NUMSTRM];
    float* l2_norm_d_res = new float[NUMSTRM];
    float* l2_norm_rand_vec_init = new float[NUMSTRM];
    thrust::fill(sum, sum+NUMSTRM, 0.0f);
    thrust::fill(tol,tol+NUMSTRM, 100.0f);
    thrust::fill(res, res+NUMSTRM*node_size, 0.0f);
    thrust::fill(vec, vec+NUMSTRM*node_size, 1.0f/node_size);
    thrust::fill(values, values+NUMSTRM*edge_size, 1.0f);
    if(!HandleCUDAError(hipMalloc((void**)&d_csc, sizeof(unsigned int)*(node_size+1)))){
        cout<<"Error allocating memory for d_csc"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_succ, sizeof(unsigned int)*(edge_size)))){
        cout<<"Error allocating memory for d_succ"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_vec, sizeof(float)*node_size*NUMSTRM))){
        cout<<"Error allocating memory for d_seed_set"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_res, sizeof(float)*node_size*NUMSTRM))){
        cout<<"Error allocating memory for d_res"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_csc, csc, sizeof(unsigned int)*node_size, hipMemcpyHostToDevice))){
        cout<<"Error copying csc to device"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_succ, succ, sizeof(unsigned int)*edge_size, hipMemcpyHostToDevice))){
        cout<<"Error copying succ to device"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_vec, vec, sizeof(float)*node_size*NUMSTRM, hipMemcpyHostToDevice))){
        cout<<"Error copying vec to device"<<endl;
    }
    delete[] vec;
    if(!HandleCUDAError(hipMemcpy(d_res, res, sizeof(float)*node_size*NUMSTRM, hipMemcpyHostToDevice))){
        cout<<"Error copying res to device"<<endl;
    }
    delete[] res;

    
    float* d_values;
    if(!HandleCUDAError(hipMalloc((void**)&d_values, sizeof(float)*(edge_size*NUMSTRM)))){
        cout<<"Error allocating memory for d_values"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_values, values, sizeof(float)*edge_size*NUMSTRM, hipMemcpyHostToDevice))){
        cout<<"Error copying values to device"<<endl;
    }
    // delete[] values;


    float* rand_init;
    if(!HandleCUDAError(hipMalloc((void**)&rand_init, NUMSTRM*num_walker*sizeof(float)))){
        std::cout<<"Error allocating memory for rand_frog"<<endl;
    }
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);

    /*Now, we have the random numbers generated*/
    float* rand_vec_init;
    float* h_rand_vec_init = new float[node_size*NUMSTRM];
    thrust::fill(h_rand_vec_init, h_rand_vec_init+node_size*NUMSTRM, 0.0f);
    if(!HandleCUDAError(hipMalloc((void**)&rand_vec_init, sizeof(float)*node_size*NUMSTRM))){
        std::cout<<"Error allocating memory for rand_vec_init"<<endl;
    } 
    if(!HandleCUDAError(hipMemcpy(rand_vec_init, h_rand_vec_init, sizeof(float)*node_size*NUMSTRM, hipMemcpyHostToDevice))){
        cout<<"Error copying h_rand_vec_init to device"<<endl;
    }

    float* store_stream_res;
    float* h_store_stream_res = new float[node_size*NUMSTRM];
    thrust::fill(h_store_stream_res, h_store_stream_res+node_size*NUMSTRM, 0.0f);
    if(!HandleCUDAError(hipMalloc((void**)&store_stream_res, sizeof(float)*node_size*NUMSTRM))){
        std::cout<<"Error allocating memory for store_stream_res"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(store_stream_res, h_store_stream_res, sizeof(float)*node_size*NUMSTRM, hipMemcpyHostToDevice))){
        cout<<"Error copying h_store_stream_res to device"<<endl;
    }
    delete[] h_store_stream_res;
    float* rand_numbers;
    if (!HandleCUDAError(hipMalloc((void**)&rand_numbers, sizeof(float) * edge_size*NUMSTRM))) {
        cout << "Error allocating memory for rand_numbers" << endl;
    }
    printCudaMemoryUsage();
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    for(int i = 0; i < epochs; i++){
        cout<<"Epoch "<<i<<endl;
        thrust::fill(tol,tol+NUMSTRM, 100.0f);
        int while_count = 0;
        while_count=0;
        srand(time(0));
        int rand_seed = rand();
        hiprandSetPseudoRandomGeneratorSeed(gen, rand_seed);
        hiprandGenerateUniform(gen, rand_init, num_walker*NUMSTRM);
        for(int i = 0; i < NUMSTRM; i++){
            //Initialize the random vector
            float* rand_init_i = rand_init + i*num_walker;
            float* rand_vec_init_i = rand_vec_init + i*node_size;
            Init_Random<<<blocks_per_stream, TPB,0,streams[i]>>>(rand_vec_init_i, rand_init_i, node_size, num_walker);
            if(!HandleCUDAError(hipStreamSynchronize(streams[i]))){
                cout<<"Error synchronizing device at Init Random for Stream "<<i<<endl;
            }

            float* rand_numbers_i = rand_numbers + i*NUMSTRM;
            float* d_values_i = d_values + i*edge_size;
            hiprandGenerator_t gen;
            hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
            srand(time(0));
            int rand_seed = rand();
            hiprandSetPseudoRandomGeneratorSeed(gen, rand_seed);
            hiprandGenerateUniform(gen, rand_numbers_i, edge_size);
            hiprandDestroyGenerator(gen);
            thrust::transform(thrust::device.on(streams[i]), rand_numbers_i, rand_numbers_i+edge_size, d_values_i, d_values_i, [threshold] __device__ (float x, float y) { return eval_values(x,y,threshold); });
        }
        while(thrust::all_of(thrust::host, tol, tol+NUMSTRM, [=] __device__ (float x) { return x > threshold; }) && while_count < 1000){
            while_count++;
            for(int i = 0; i < NUMSTRM; i++){
                //Perform the first iteration of the algorithm
                if(tol[i] > threshold){
                    float* rand_vec_init_i = rand_vec_init + i*node_size;
                    float* d_res_i = d_res + i*node_size;
                    float* d_values_i = d_values + i*edge_size;
                    sparseCSRMat_Vec_Mult<<<blocks_per_stream, TPB,0,streams[i]>>>(d_csc, d_succ, d_values_i, rand_vec_init_i, d_res_i, node_size);  
                    if(!HandleCUDAError(hipStreamSynchronize(streams[i]))){
                        cout<<"Error synchronizing device at sparseCSRMat_Vec_Mult for stream "<<i<<endl;
                    }
                }
            }
            for(int i = 0; i < NUMSTRM; i++){
                // Add 1/n to the vector
                if(tol[i] > threshold){
                    float* d_res_i = d_res + i*node_size;
                    float* d_vec_i = d_vec + i*node_size;
                    float* rand_vec_init_i = rand_vec_init + i*node_size;
                    float* store_stream_res_i = store_stream_res + i*node_size;
                    Float_VectAdd<<<blocks_per_stream, TPB,0,streams[i]>>>(d_res_i,d_vec_i, node_size);
                    if(!HandleCUDAError(hipStreamSynchronize(streams[i]))){
                        cout<<"Error synchronizing device for Float_VectAdd at stream "<<i<<endl;
                    }
                    //Need to normalize the vector using thrust library

                    l2_norm_d_res[i] = thrust::transform_reduce(thrust::device, d_res_i, d_res_i + node_size, [] __device__ (float x) { return x * x; }, 0.0f, thrust::plus<float>());
                    l2_norm_d_res[i] = sqrt(l2_norm_d_res[i]);

                    l2_norm_rand_vec_init[i] = thrust::transform_reduce(thrust::device, rand_vec_init_i, rand_vec_init_i + node_size, [] __device__ (float x) { return x * x; }, 0.0f, thrust::plus<float>());
                    l2_norm_rand_vec_init[i] = sqrt(l2_norm_rand_vec_init[i]);

                    tol[i] = abs(l2_norm_d_res[i]-l2_norm_rand_vec_init[i]);
                    thrust::copy(thrust::device.on(streams[i]), d_res_i, d_res_i+node_size, rand_vec_init_i);


                    sum[i] = thrust::reduce(thrust::device.on(streams[i]), rand_vec_init_i, rand_vec_init_i+node_size);
                    float temp = sum[i];
                    thrust::transform(thrust::device.on(streams[i]), rand_vec_init_i, rand_vec_init_i+node_size, rand_vec_init_i, [=] __device__ (float x) { return x/temp; });
                    Float_VectAdd<<<blocks_per_stream, TPB,0,streams[i]>>>(store_stream_res_i,rand_vec_init_i, node_size);
                    if(!HandleCUDAError(hipStreamSynchronize(streams[i]))){
                        cout<<"Error synchronizing device for Float_VectAdd at stream "<<i<<endl;
                    }
                    // thrust::fill(thrust::device.on(streams[i]), d_vec, d_vec+node_size, 1.0f/node_size);
                }
            }
        }
        for(int i = 0; i<NUMSTRM;i++){
            thrust::fill(thrust::device.on(streams[i]), d_values+i*edge_size, d_values+(i+1)*edge_size, 1.0f);
        }
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hiprandDestroyGenerator(gen);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    cout<<"Time taken: "<<milliseconds<<endl;
    Save_Data(file, milliseconds, damping_factor, threshold, epochs);
    if(!HandleCUDAError(hipFree(d_csc))){
        cout<<"Error freeing d_csc"<<endl;
    }
    if(!HandleCUDAError(hipFree(d_succ))){
        cout<<"Error freeing d_succ"<<endl;
    }
    if(!HandleCUDAError(hipFree(d_vec))){
        cout<<"Error freeing d_vec"<<endl;
    }
    if(!HandleCUDAError(hipFree(d_res))){
        cout<<"Error freeing d_res"<<endl;
    }
    if(!HandleCUDAError(hipFree(d_values))){
        cout<<"Error freeing d_values"<<endl;
    }
    unsigned int* rand_idx;
    unsigned int* h_rand_idx = new unsigned int[node_size];
    thrust::fill(h_rand_idx, h_rand_idx+node_size, 0);
    if(!HandleCUDAError(hipMalloc((void**)&rand_idx, sizeof(unsigned int)*node_size))){
        cout<<"Error allocating memory for rand_idx"<<endl;
    }
    thrust::sequence(thrust::device, rand_idx, rand_idx+node_size);
    //Take the sum of the vectors and then sort them
    for(int i = 1; i<NUMSTRM;i++){
        thrust::transform(thrust::device, store_stream_res, store_stream_res+node_size, store_stream_res+i*node_size, store_stream_res, thrust::plus<float>());
    }
    thrust::sort_by_key(thrust::device, store_stream_res, store_stream_res+node_size, rand_idx, thrust::greater<float>());
    //Get the top k indexes
    if(!HandleCUDAError(hipMemcpy(h_rand_idx, rand_idx, sizeof(unsigned int)*K, hipMemcpyDeviceToHost))){
        cout<<"Error copying rand_idx to host"<<endl;
    }
    for(int i = 0; i < K; i++){
        seed_set[i] = h_rand_idx[i];
    }
    if(!HandleCUDAError(hipFree(rand_idx))){
        cout<<"Error freeing rand_idx"<<endl;
    }
    delete[] h_rand_idx;

    for(int i = 0; i<NUMSTRM;i++){
        if(!HandleCUDAError(hipStreamDestroy(streams[i]))){
            cout<<"Error destroying stream number "<<i<<endl;
        }
    }
    if(!HandleCUDAError(hipFree(store_stream_res))){
        cout<<"Error freeing store_stream_res"<<endl;
    }
    if(!HandleCUDAError(hipFree(rand_numbers))){
        cout<<"Error freeing rand_numbers"<<endl;
    }
    if(!HandleCUDAError(hipFree(rand_vec_init))){
        cout<<"Error freeing rand_vec_init"<<endl;
    }
    if(!HandleCUDAError(hipFree(rand_init))){
        cout<<"Error freeing rand_init"<<endl;
    }
    delete[] h_rand_vec_init;
    delete[] l2_norm_rand_vec_init;
    delete[] l2_norm_d_res;
    delete[] sum;
    delete[] tol;
    delete[] values;
}


__host__ void  RIM_rand_Ver3_PR(unsigned int* csc, unsigned int* succ, unsigned int node_size, unsigned int edge_size, unsigned int* seed_set, string file){
    float threshold = .8;
    float damping_factor =.3;
    hipDeviceProp_t prop;
    int device;
    hipGetDevice(&device);  // Get the current device
    hipGetDeviceProperties(&prop, device);  // Get the properties of the device

    int maxActiveBlocksPerMultiprocessor = prop.maxThreadsPerMultiProcessor / TPB;
    int maxActiveBlocks = prop.multiProcessorCount * maxActiveBlocksPerMultiprocessor;
    int blocks_per_stream = maxActiveBlocks/NUMSTRM+1;

    printf("Max active blocks: %d\n", maxActiveBlocks);
    hipStream_t* streams = (hipStream_t*)malloc(sizeof(hipStream_t)*NUMSTRM);
    for(int i = 0; i < NUMSTRM; i++){
        if(!HandleCUDAError(hipStreamCreate(&streams[i]))){
            cout<<"Error creating stream number "<<i<<endl;
        }
    }
    float* rand_vec_init;
    float* h_rand_vec_init = new float[node_size*NUMSTRM];
    float* pr_vector = new float[node_size];
    float* pr_time = new float[1];
    *pr_time = 0.0f;
    thrust::fill(h_rand_vec_init, h_rand_vec_init+node_size*NUMSTRM, 0.0f);
    thrust::fill(pr_vector, pr_vector+node_size, 0.0f);
    PageRank(pr_vector,csc,succ,.15f,node_size,edge_size,100,1e-6,pr_time);
    float* d_pr;
    if(!HandleCUDAError(hipMalloc((void**)&d_pr, sizeof(float)*node_size))){
        cout<<"Error allocating memory for d_pr"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_pr, pr_vector, sizeof(float)*node_size, hipMemcpyHostToDevice))){
        cout<<"Error copying pr_vector to device"<<endl;
    }
    delete[] pr_vector;
    unsigned int epochs=30;
    unsigned int* d_csc;
    unsigned int* d_succ;
    float* d_vec; //we will use the seed set as the PR vector and then transfer the top k to the actual seed set
    float* d_res;
    float* vec = new float[NUMSTRM*node_size];
    float* values = new float[NUMSTRM*edge_size];
    float* res = new float[NUMSTRM*node_size]; 
    float* tol = new float[NUMSTRM];
    float* sum = new float[NUMSTRM];
    float* l2_norm_d_res = new float[NUMSTRM];
    float* l2_norm_rand_vec_init = new float[NUMSTRM];
    thrust::fill(sum, sum+NUMSTRM, 0.0f);
    thrust::fill(tol,tol+NUMSTRM, 100.0f);
    thrust::fill(res, res+NUMSTRM*node_size, 0.0f);
    thrust::fill(vec, vec+NUMSTRM*node_size, 1.0f/node_size);
    thrust::fill(values, values+NUMSTRM*edge_size, 1.0f);
    if(!HandleCUDAError(hipMalloc((void**)&d_csc, sizeof(unsigned int)*(node_size+1)))){
        cout<<"Error allocating memory for d_csc"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_succ, sizeof(unsigned int)*(edge_size)))){
        cout<<"Error allocating memory for d_succ"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_vec, sizeof(float)*node_size*NUMSTRM))){
        cout<<"Error allocating memory for d_seed_set"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_res, sizeof(float)*node_size*NUMSTRM))){
        cout<<"Error allocating memory for d_res"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_csc, csc, sizeof(unsigned int)*node_size, hipMemcpyHostToDevice))){
        cout<<"Error copying csc to device"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_succ, succ, sizeof(unsigned int)*edge_size, hipMemcpyHostToDevice))){
        cout<<"Error copying succ to device"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_vec, vec, sizeof(float)*node_size*NUMSTRM, hipMemcpyHostToDevice))){
        cout<<"Error copying vec to device"<<endl;
    }
    delete[] vec;
    if(!HandleCUDAError(hipMemcpy(d_res, res, sizeof(float)*node_size*NUMSTRM, hipMemcpyHostToDevice))){
        cout<<"Error copying res to device"<<endl;
    }
    delete[] res;

    
    float* d_values;
    if(!HandleCUDAError(hipMalloc((void**)&d_values, sizeof(float)*(edge_size*NUMSTRM)))){
        cout<<"Error allocating memory for d_values"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_values, values, sizeof(float)*edge_size*NUMSTRM, hipMemcpyHostToDevice))){
        cout<<"Error copying values to device"<<endl;
    }
    // delete[] values;

    if(!HandleCUDAError(hipMalloc((void**)&rand_vec_init, sizeof(float)*node_size*NUMSTRM))){
        std::cout<<"Error allocating memory for rand_vec_init"<<endl;
    } 
    if(!HandleCUDAError(hipMemcpy(rand_vec_init, h_rand_vec_init, sizeof(float)*node_size*NUMSTRM, hipMemcpyHostToDevice))){
        cout<<"Error copying h_rand_vec_init to device"<<endl;
    }

    float* store_stream_res;
    float* h_store_stream_res = new float[node_size*NUMSTRM];
    thrust::fill(h_store_stream_res, h_store_stream_res+node_size*NUMSTRM, 0.0f);
    if(!HandleCUDAError(hipMalloc((void**)&store_stream_res, sizeof(float)*node_size*NUMSTRM))){
        std::cout<<"Error allocating memory for store_stream_res"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(store_stream_res, h_store_stream_res, sizeof(float)*node_size*NUMSTRM, hipMemcpyHostToDevice))){
        cout<<"Error copying h_store_stream_res to device"<<endl;
    }
    delete[] h_store_stream_res;
    float* rand_numbers;
    if (!HandleCUDAError(hipMalloc((void**)&rand_numbers, sizeof(float) * edge_size*NUMSTRM))) {
        cout << "Error allocating memory for rand_numbers" << endl;
    }
    printCudaMemoryUsage();
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    for(int i = 0; i < epochs; i++){
        cout<<"Epoch "<<i<<endl;
        thrust::fill(tol,tol+NUMSTRM, 100.0f);
        int while_count = 0;
        while_count=0;
        for(int i =0; i<NUMSTRM;i++){
            thrust::copy(thrust::device.on(streams[i]), d_pr, d_pr+node_size, rand_vec_init+i*node_size);
            //Initialize the random vector
            float* rand_numbers_i = rand_numbers + i*NUMSTRM;
            float* d_values_i = d_values + i*edge_size;
            hiprandGenerator_t gen;
            hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
            srand(time(0));
            int rand_seed = rand();
            hiprandSetPseudoRandomGeneratorSeed(gen, rand_seed);
            hiprandGenerateUniform(gen, rand_numbers_i, edge_size);
            hiprandDestroyGenerator(gen);
            thrust::transform(thrust::device.on(streams[i]), rand_numbers_i, rand_numbers_i+edge_size, d_values_i, d_values_i, [threshold] __device__ (float x, float y) { return eval_values(x,y,threshold); });
        }
        while(thrust::all_of(thrust::host, tol, tol+NUMSTRM, [=] __device__ (float x) { return x > threshold; }) && while_count < 1000){
            while_count++;
            for(int i = 0; i < NUMSTRM; i++){
                //Perform the first iteration of the algorithm
                if(tol[i] > threshold){
                    float* rand_numbers_i = rand_numbers + i*NUMSTRM;
                    float* rand_vec_init_i = rand_vec_init + i*node_size;
                    float* d_res_i = d_res + i*node_size;
                    float* d_values_i = d_values + i*edge_size;
                    sparseCSRMat_Vec_Mult<<<blocks_per_stream, TPB,0,streams[i]>>>(d_csc, d_succ, d_values_i, rand_vec_init_i, d_res_i, node_size);  
                    if(!HandleCUDAError(hipStreamSynchronize(streams[i]))){
                        cout<<"Error synchronizing device at sparseCSRMat_Vec_Mult for stream "<<i<<endl;
                    }
                }
            }
            for(int i = 0; i < NUMSTRM; i++){
                // Add 1/n to the vector
                if(tol[i] > threshold){
                    float* d_res_i = d_res + i*node_size;
                    float* d_vec_i = d_vec + i*node_size;
                    float* rand_vec_init_i = rand_vec_init + i*node_size;
                    float* store_stream_res_i = store_stream_res + i*node_size;
                    Float_VectAdd<<<blocks_per_stream, TPB,0,streams[i]>>>(d_res_i,d_vec_i, node_size);
                    if(!HandleCUDAError(hipStreamSynchronize(streams[i]))){
                        cout<<"Error synchronizing device for Float_VectAdd at stream "<<i<<endl;
                    }
                    //Need to normalize the vector using thrust library

                    l2_norm_d_res[i] = thrust::transform_reduce(thrust::device, d_res_i, d_res_i + node_size, [] __device__ (float x) { return x * x; }, 0.0f, thrust::plus<float>());
                    l2_norm_d_res[i] = sqrt(l2_norm_d_res[i]);

                    l2_norm_rand_vec_init[i] = thrust::transform_reduce(thrust::device, rand_vec_init_i, rand_vec_init_i + node_size, [] __device__ (float x) { return x * x; }, 0.0f, thrust::plus<float>());
                    l2_norm_rand_vec_init[i] = sqrt(l2_norm_rand_vec_init[i]);

                    tol[i] = abs(l2_norm_d_res[i]-l2_norm_rand_vec_init[i]);
                    thrust::copy(thrust::device.on(streams[i]), d_res_i, d_res_i+node_size, rand_vec_init_i);


                    sum[i] = thrust::reduce(thrust::device.on(streams[i]), rand_vec_init_i, rand_vec_init_i+node_size);
                    float temp = sum[i];
                    thrust::transform(thrust::device.on(streams[i]), rand_vec_init_i, rand_vec_init_i+node_size, rand_vec_init_i, [=] __device__ (float x) { return x/temp; });
                    Float_VectAdd<<<blocks_per_stream, TPB,0,streams[i]>>>(store_stream_res_i,rand_vec_init, node_size);
                    if(!HandleCUDAError(hipStreamSynchronize(streams[i]))){
                        cout<<"Error synchronizing device for Float_VectAdd at stream "<<i<<endl;
                    }
                    // thrust::fill(thrust::device.on(streams[i]), d_vec, d_vec+node_size, 1.0f/node_size);
                }
            }
        }
        for(int i = 0; i<NUMSTRM;i++){
            thrust::fill(thrust::device.on(streams[i]), d_values+i*edge_size, d_values+(i+1)*edge_size, 1.0f);
        }
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    cout<<"Time taken: "<<milliseconds<<endl;
    Save_Data(file, milliseconds, damping_factor, threshold, epochs);
    if(!HandleCUDAError(hipFree(d_csc))){
        cout<<"Error freeing d_csc"<<endl;
    }
    if(!HandleCUDAError(hipFree(d_succ))){
        cout<<"Error freeing d_succ"<<endl;
    }
    if(!HandleCUDAError(hipFree(d_vec))){
        cout<<"Error freeing d_vec"<<endl;
    }
    if(!HandleCUDAError(hipFree(d_res))){
        cout<<"Error freeing d_res"<<endl;
    }
    if(!HandleCUDAError(hipFree(d_values))){
        cout<<"Error freeing d_values"<<endl;
    }
    unsigned int* rand_idx;
    unsigned int* h_rand_idx = new unsigned int[node_size];
    thrust::fill(h_rand_idx, h_rand_idx+node_size, 0);
    if(!HandleCUDAError(hipMalloc((void**)&rand_idx, sizeof(unsigned int)*node_size))){
        cout<<"Error allocating memory for rand_idx"<<endl;
    }
    thrust::sequence(thrust::device, rand_idx, rand_idx+node_size);
    //Take the sum of the vectors and then sort them
    for(int i = 1; i<NUMSTRM;i++){
        thrust::transform(thrust::device, store_stream_res, store_stream_res+node_size, store_stream_res+i*node_size, store_stream_res, thrust::plus<float>());
    }
    thrust::sort_by_key(thrust::device, store_stream_res, store_stream_res+node_size, rand_idx, thrust::greater<float>());
    //Get the top k indexes
    if(!HandleCUDAError(hipMemcpy(h_rand_idx, rand_idx, sizeof(unsigned int)*K, hipMemcpyDeviceToHost))){
        cout<<"Error copying rand_idx to host"<<endl;
    }
    for(int i = 0; i < K; i++){
        seed_set[i] = h_rand_idx[i];
    }
    if(!HandleCUDAError(hipFree(rand_idx))){
        cout<<"Error freeing rand_idx"<<endl;
    }
    delete[] h_rand_idx;

    for(int i = 0; i<NUMSTRM;i++){
        if(!HandleCUDAError(hipStreamDestroy(streams[i]))){
            cout<<"Error destroying stream number "<<i<<endl;
        }
    }
    if(!HandleCUDAError(hipFree(store_stream_res))){
        cout<<"Error freeing store_stream_res"<<endl;
    }
    if(!HandleCUDAError(hipFree(rand_numbers))){
        cout<<"Error freeing rand_numbers"<<endl;
    }
    if(!HandleCUDAError(hipFree(rand_vec_init))){
        cout<<"Error freeing rand_vec_init"<<endl;
    }
    delete[] h_rand_vec_init;
    delete[] l2_norm_rand_vec_init;
    delete[] l2_norm_d_res;
    delete[] sum;
    delete[] tol;
    delete[] values;
}



__host__ void CheckSparseMatVec(unsigned int* csc, unsigned int* succ,edge* edge_list, unsigned int node_size, unsigned int edge_size){
    unsigned int* d_csc;
    unsigned int* d_succ;
    float* d_vec; //we will use the seed set as the PR vector and then transfer the top k to the actual seed set
    float* d_res;
    float* vec = new float[node_size];
    float* values = new float[edge_size];
    float* res = new float[node_size]; 
    thrust::fill(res, res+node_size, 0.0f);
    thrust::fill(vec, vec+node_size, 1.0f/node_size);
    thrust::fill(values, values+edge_size, 1.0f);
    hipDeviceProp_t prop;
    int device;
    hipGetDevice(&device);  // Get the current device
    hipGetDeviceProperties(&prop, device);  // Get the properties of the device

    int maxActiveBlocksPerMultiprocessor = prop.maxThreadsPerMultiProcessor / TPB;
    int maxActiveBlocks = prop.multiProcessorCount * maxActiveBlocksPerMultiprocessor;
    int blocks_per_stream = maxActiveBlocks/NUMSTRM;

    printf("Max active blocks: %d\n", maxActiveBlocks);
    hipStream_t* streams = (hipStream_t*)malloc(sizeof(hipStream_t)*NUMSTRM);
    for(int i = 0; i < NUMSTRM; i++){
        if(!HandleCUDAError(hipStreamCreate(&streams[i]))){
            cout<<"Error creating stream number "<<i<<endl;
        }
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_csc, sizeof(unsigned int)*(node_size+1)))){
        cout<<"Error allocating memory for d_csc"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_succ, sizeof(unsigned int)*(edge_size)))){
        cout<<"Error allocating memory for d_succ"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_vec, sizeof(float)*node_size))){
        cout<<"Error allocating memory for d_seed_set"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_res, sizeof(float)*node_size))){
        cout<<"Error allocating memory for d_res"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_csc, csc, sizeof(unsigned int)*node_size, hipMemcpyHostToDevice))){
        cout<<"Error copying csc to device"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_succ, succ, sizeof(unsigned int)*edge_size, hipMemcpyHostToDevice))){
        cout<<"Error copying succ to device"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_vec, vec, sizeof(float)*node_size, hipMemcpyHostToDevice))){
        cout<<"Error copying vec to device"<<endl;
    }
    delete[] vec;
    if(!HandleCUDAError(hipMemcpy(d_res, res, sizeof(float)*node_size, hipMemcpyHostToDevice))){
        cout<<"Error copying res to device"<<endl;
    }
    delete[] res;
    
    float* d_values;
    if(!HandleCUDAError(hipMalloc((void**)&d_values, sizeof(float)*(edge_size)))){
        cout<<"Error allocating memory for d_values"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_values, values, sizeof(float)*edge_size, hipMemcpyHostToDevice))){
        cout<<"Error copying values to device"<<endl;
    }
    // delete[] values;
    // if(!HandleCUDAError(hipMemcpy(values,d_values, sizeof(float)*edge_size, hipMemcpyDeviceToHost))){
    //     cout<<"Error copying values to device"<<endl;
    // }

    float* rand_init;
    if(!HandleCUDAError(hipMalloc((void**)&rand_init, K*sizeof(float)))){
        std::cout<<"Error allocating memory for rand_frog"<<endl;
    }
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    srand(time(0));
    int rand_seed = rand();
    hiprandSetPseudoRandomGeneratorSeed(gen, rand_seed);
    hiprandGenerateUniform(gen, rand_init, K);
    /*Now, we have the random numbers generated*/
    hiprandDestroyGenerator(gen);
    float* rand_vec_init;
    float* h_rand_vec_init = new float[node_size]{0.0f};
    if(!HandleCUDAError(hipMalloc((void**)&rand_vec_init, sizeof(float)*node_size))){
        std::cout<<"Error allocating memory for rand_vec_init"<<endl;
    } 
    if(!HandleCUDAError(hipMemcpy(rand_vec_init, h_rand_vec_init, sizeof(float)*node_size, hipMemcpyHostToDevice))){
        cout<<"Error copying h_rand_vec_init to device"<<endl;
    }
    //Initialize the random vector
    Init_Random<<<blocks_per_stream, TPB>>>(rand_vec_init, rand_init, node_size, K);
    if(!HandleCUDAError(hipDeviceSynchronize())){
        cout<<"Error synchronizing device"<<endl;
    }
    //Perform the first iteration of the algorithm
    sparseCSRMat_Vec_Mult<<<blocks_per_stream, TPB>>>(d_csc, d_succ, d_values, rand_vec_init, d_res, node_size);  
    if(!HandleCUDAError(hipDeviceSynchronize())){
        cout<<"Error synchronizing device"<<endl;
    }
    Float_VectAdd<<<blocks_per_stream, TPB>>>(d_res, d_vec, node_size);
    if(!HandleCUDAError(hipDeviceSynchronize())){
        cout<<"Error synchronizing device"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(h_rand_vec_init, rand_vec_init, sizeof(float)*node_size, hipMemcpyDeviceToHost))){
        cout<<"Error copying d_vec to host"<<endl;
    }
    float* h_res_GPU = new float[node_size]{0.0f};

    if(!HandleCUDAError(hipMemcpy(h_res_GPU, d_res, sizeof(float)*node_size, hipMemcpyDeviceToHost))){
        cout<<"Error copying d_vec to host"<<endl;
    }

    float* h_res_CPU = new float[node_size]{0.0f};
    float* A = (float*)malloc(sizeof(float)*node_size*node_size);
    GenAdj(edge_list, A, node_size, edge_size);
    h_MatVecMult(A, h_rand_vec_init, h_res_CPU, node_size);
    float* support_vec = new float[node_size];
    thrust::fill(support_vec, support_vec+node_size, 1.0f/node_size);
    for(int i = 0; i < node_size; i++){
        h_res_CPU[i] += support_vec[i];
    }
    float sum = 0.0f;
    sum = thrust::inner_product(thrust::device, d_res, d_res+node_size, d_res, 0.0f);
    sum = sqrt(sum);
    thrust::transform(thrust::device, d_res, d_res+node_size, d_res, thrust::placeholders::_1/sum);
    if(!HandleCUDAError(hipDeviceSynchronize())){
        cout<<"Error synchronizing device"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(h_res_GPU, d_res, sizeof(float)*node_size, hipMemcpyDeviceToHost))){
        cout<<"Error copying d_vec to host"<<endl;
    }
    Normalize_L2(h_res_CPU, node_size);
    Verify(h_res_GPU, h_res_CPU, node_size);

}

__global__ void sparseCSRMat_Vec_Mult(unsigned int* csc, unsigned int* succ, float* values, float* vec, float* result, unsigned int node_size){
    unsigned int tid = threadIdx.x + blockIdx.x*blockDim.x;
    for(int t = tid; t < node_size; t+=blockDim.x*gridDim.x){
        unsigned int start = csc[t];
        unsigned int end = csc[t+1];
        float sum = 0.0f;
        for(int i = start; i < end; i++){
            sum += values[i]*vec[succ[i]];
        }
        result[t] = sum;
    }
}

__global__ void Float_VectAdd(float* vec1, float* vec2, unsigned int size){
    unsigned int tid = threadIdx.x + blockIdx.x*blockDim.x;
    if(tid < size){
        vec1[tid] += vec2[tid];
    }
}

__global__ void Init_Random(float* vec, float* rand_init, unsigned int size, unsigned int k){
    unsigned int tid = threadIdx.x + blockIdx.x*blockDim.x;
    unsigned int idx =0;
    if(tid<k){
        rand_init[tid] = floorf(rand_init[tid]*size);
        idx = (unsigned int)rand_init[tid];
        idx = idx%size;
        vec[idx] = 1.0f;
    }
}

__host__ void Verify(float* gpu_vec, float* cpu_vec, unsigned int size){
    float sum = 0.0f;
    for(int i = 0; i < size; i++){
        sum = abs(gpu_vec[i]-cpu_vec[i]);
        if (sum > 0.00001f){
            cout<<"Error at index "<<i<<endl;
            cout<<"GPU: "<<gpu_vec[i]<<endl;
            cout<<"CPU: "<<cpu_vec[i]<<endl;
            return;
        }
    }
    cout<<"No errors found"<<endl;
}