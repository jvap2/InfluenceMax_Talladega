#include "hip/hip_runtime.h"
#include "../include/data.h"
/*The code for this come from Tang et al IMM Algorithm
The psuedo code is as follows:
1) Initialize a set R={empty set} and an integer LB = 1
2) Let epsilon'=sqrt(2)epsilon
3) for i = 1 to log_2(n)-1 do:
4) 	Let x=n/2^i
5) 	Let theta = lambda'/x
6)  while |R|<theta_i do:
7)      Select a node from G uniformly at random
8)      Generate an RR set for v, and insert it into R
9)  Let S_i = NodeSelection(R)
10) if n*FR(S)>=(1+epsilon')*x then:
11)     LB = n*FR(S)/(1+epsilon')
12)     break
13) Let theta= lambda^{*}/LB
14) while |R|<theta do:
15)     Select a node from G uniformly at random
16)     Generate an RR set for v, and insert it into R
17) Return R


What is going to be needed:
1) A graph
2) A set of RR sets
3) A set of nodes
4) A set of edges

How do we want to store these:
1) The RR sets would be traversable with CSC format, and any forward traversal will need
but would it be convenient to traverse both ways with a COO format?
2) We may be able to use linked lists as well but this will be slower, but also harder to implement, and harder for generating RRR sets

*/


__host__ void  RIM_rand_Ver1(unsigned int* csc, unsigned int* succ, unsigned int node_size, unsigned int edge_size, unsigned int* seed_set){
    float threshold = 0.75;
    hipStream_t* streams = (hipStream_t*)malloc(sizeof(hipStream_t)*NUMSTRM);
    for(int i = 0; i < NUMSTRM; i++){
        if(!HandleCUDAError(hipStreamCreate(&streams[i]))){
            cout<<"Error creating stream number "<<i<<endl;
        }
    }
    unsigned int* d_csc;
    unsigned int* d_succ;
    float* d_vec; //we will use the seed set as the PR vector and then transfer the top k to the actual seed set
    if(!HandleCUDAError(hipMalloc((void**)&d_csc, sizeof(unsigned int)*node_size))){
        cout<<"Error allocating memory for d_csc"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_succ, sizeof(unsigned int)*(edge_size+1)))){
        cout<<"Error allocating memory for d_succ"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_vec, sizeof(float)*node_size))){
        cout<<"Error allocating memory for d_seed_set"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_csc, csc, sizeof(unsigned int)*node_size, hipMemcpyHostToDevice))){
        cout<<"Error copying csc to device"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_succ, succ, sizeof(unsigned int)*edge_size, hipMemcpyHostToDevice))){
        cout<<"Error copying succ to device"<<endl;
    }
    if(!HandleCUDAError(hipMemset(d_vec, 1.0f/node_size, sizeof(float)*node_size))){
        cout<<"Error setting d_vec to 1/n"<<endl;
    }
    
    float* d_values;
    if(!HandleCUDAError(hipMalloc((void**)&d_values, sizeof(float)*(edge_size+1)))){
        cout<<"Error allocating memory for d_values"<<endl;
    }
    if(!HandleCUDAError(hipMemset(d_values, 1.0f, sizeof(float)*(edge_size+1)))){
        cout<<"Error setting d_values to 1"<<endl;
    }
    unsigned int num_blocks = (node_size+TPB-1)/TPB;
    unsigned int num_blocks2 = (edge_size+TPB-1)/TPB;
    float* rand_init;
    if(!HandleCUDAError(hipMalloc((void**)&rand_init, K*sizeof(float)))){
        std::cout<<"Error allocating memory for rand_frog"<<endl;
    }
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    srand(time(0));
    int rand_seed = rand();
    hiprandSetPseudoRandomGeneratorSeed(gen, rand_seed);
    hiprandGenerateUniform(gen, rand_init, K);
    /*Now, we have the random numbers generated*/
    hiprandDestroyGenerator(gen);
    float* rand_vec_init;
    if(!HandleCUDAError(hipMalloc((void**)&rand_vec_init, sizeof(float)*node_size))){
        std::cout<<"Error allocating memory for rand_vec_init"<<endl;
    } 
    sparseCSRMat_Vec_Mult<<<num_blocks2,TPB>>>(d_csc, d_succ, d_values,d_vec, node_size);


    
}


__global__ void sparseCSRMat_Vec_Mult(unsigned int* csc, unsigned int* succ, float* vec, float* result, unsigned int node_size){
    unsigned int tid = threadIdx.x + blockIdx.x*blockDim.x;
    for(int t = tid; t < node_size; t+=blockDim.x*gridDim.x){
        unsigned int start = csc[t];
        unsigned int end = csc[t+1];
        unsigned int sum = 0;
        for(int i = start; i < end; i++){
            sum += vec[succ[i]];
        }
        result[t] = sum;
    }
}

__global__ void Float_VectAdd(float* vec1, float* vec2, unsigned int size){
    unsigned int tid = threadIdx.x + blockIdx.x*blockDim.x;
    if(tid < size){
        vec1[tid] += vec2[tid];
    }
}

__global__ void Init_Random(float* vec, float* rand_init, unsigned int size, unsigned int k){
    unsigned int tid = threadIdx.x + blockIdx.x*blockDim.x;
    if(tid<k){
        unsigned int idx = (unsigned int)(rand_init[tid]*size);
        idx = idx%size;
        vec[idx] = 1.0f;
    }
}